#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
	 LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
	 http://lammps.sandia.gov, Sandia National Laboratories
	 Steve Plimpton, sjplimp@sandia.gov

	 Copyright (2003) Sandia Corporation.	Under the terms of Contract
	 DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
	 certain rights in this software.	This software is distributed under 
	 the GNU General Public License.

	 See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

#include "fix_mui_meso.h"

#include "mpi.h"
#include "stdio.h"
#include "string.h"
#include "error.h"

#include "atom_vec_meso.h"
#include "engine_meso.h"
#include "atom_meso.h"
#include "comm_meso.h"

#include "atom.h"
#include "comm.h"
#include "domain.h"
#include "force.h"
#include "input.h"
#include "region.h"
#include "update.h"
#include "variable.h"

#include "mui/mui.h"

using namespace LAMMPS_NS;
using namespace FixConst;
using namespace std;

__global__ void gpu_push_gather(
	double4* __restrict push_buffer,
	uint* __restrict push_count,
	r64* __restrict coord_x,
	r64* __restrict coord_y,
	r64* __restrict coord_z,
	r64* __restrict veloc_x,
	r64* __restrict veloc_y,
	r64* __restrict veloc_z,
	int* __restrict mask,
	const r64 push_upper,
	const r64 push_lower,
	const int  groupbit,
	const int  n_atom )
{
	for(int i = blockDim.x * blockIdx.x + threadIdx.x ; i < n_atom ; i += gridDim.x * blockDim.x ) {
		if ( ( mask[i] & groupbit ) && coord_z[i] >= push_lower && coord_z[i] <= push_upper ) {
			uint p = atomicInc( push_count, 0xFFFFFFFF );
			double4 info;
			info.x = coord_x[i];
			info.y = coord_y[i];
			info.z = coord_z[i];
			info.w = veloc_x[i];
			push_buffer[p] = info;
		}
	}
}

vector<double4> FixMuiMeso::gpu_push() {
	static int2 grid_cfg;
	static HostScalar<double4> hst_push_buffer(this->lmp,"FixMUI::push_buffer");
	static DeviceScalar<uint>  dev_push_count (this->lmp,"FixMUI::push_count");

	if ( !grid_cfg.x )
	{
		grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_push_gather, 0, hipFuncCachePreferL1 );
		hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_push_gather), hipFuncCachePreferL1 );
		dev_push_count.grow(1);
	}
	if ( hst_push_buffer.n_elem() < atom->nlocal ) {
		hst_push_buffer.grow( atom->nlocal );
	}

	Region *region = domain->regions[ipush_region];
	push_lower = region->extent_zlo;
	push_upper = region->extent_zhi;

	dev_push_count.set( 0, meso_device->stream() );
	gpu_push_gather<<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>>(
		hst_push_buffer,
		dev_push_count,
		meso_atom->dev_coord(0),
		meso_atom->dev_coord(1),
		meso_atom->dev_coord(2),
		meso_atom->dev_veloc(0),
		meso_atom->dev_veloc(1),
		meso_atom->dev_veloc(2),
		meso_atom->dev_mask,
		push_upper,
		push_lower,
		groupbit,
		atom->nlocal );

	uint n;
	dev_push_count.download( &n, 1 );
	meso_device->sync_device();
	vector<double4> result;
	for(int i=0;i<n;i++) result.push_back(hst_push_buffer[i]);
	return result;
}

__global__ void gpu_fetch_pred(
	int* __restrict pred,
	double4* __restrict loc,
	r64* __restrict coord_x,
	r64* __restrict coord_y,
	r64* __restrict coord_z,
	int* __restrict mask,
	const r64 fetch_upper,
	const r64 fetch_lower,
	const int  groupbit,
	const int  n_atom )
{
	for(int i = blockDim.x * blockIdx.x + threadIdx.x ; i < n_atom ; i += gridDim.x * blockDim.x ) {
		if ( ( mask[i] & groupbit ) && coord_z[i] >= fetch_lower && coord_z[i] <= fetch_upper ) {
			pred[i] = 1;
			loc[i].x = coord_x[i];
			loc[i].y = coord_y[i];
			loc[i].z = coord_z[i];
		}
		else
			pred[i] = 0;
	}
}

pair<vector<int>, vector<double4> > FixMuiMeso::gpu_fetch_predicate() {
	static int2 grid_cfg;
	static HostScalar<int>     hst_fetch_pred(this->lmp,"FixMUI::fetch_pred");
	static HostScalar<double4> hst_fetch_loc(this->lmp,"FixMUI::fetch_coord");
	static vector<int> host_buffer;

	if ( !grid_cfg.x )
	{
		grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_fetch_pred, 0, hipFuncCachePreferL1 );
		hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_fetch_pred), hipFuncCachePreferL1 );
	}
	if ( hst_fetch_pred.n_elem() < atom->nlocal ) {
		hst_fetch_pred.grow( atom->nlocal );
		hst_fetch_loc.grow( atom->nlocal );
	}


	Region *region = domain->regions[ifetch_region];
	fetch_lower = region->extent_zlo;
	fetch_upper = region->extent_zhi;

	gpu_fetch_pred<<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>>(
		hst_fetch_pred,
		hst_fetch_loc,
		meso_atom->dev_coord(0),
		meso_atom->dev_coord(1),
		meso_atom->dev_coord(2),
		meso_atom->dev_mask,
		fetch_upper,
		fetch_lower,
		groupbit,
		atom->nlocal );

	meso_device->sync_device();
	vector<int> result_first;
	vector<double4> result_second;
	for(int i=0;i<hst_fetch_pred.n_elem();i++) {
		result_first.push_back( hst_fetch_pred[i] );
		result_second.push_back( hst_fetch_loc[i] );
	}
	return make_pair(result_first,result_second);
}

__global__ void gpu_scatter_fetch(
	int* __restrict pred,
	double* __restrict vres,
	r64* __restrict veloc_x,
	r64* __restrict veloc_y,
	r64* __restrict veloc_z,
	int* __restrict mask,
	const int  groupbit,
	const int  n_atom )
{
	for(int i = blockDim.x * blockIdx.x + threadIdx.x ; i < n_atom ; i += gridDim.x * blockDim.x ) {
		if ( pred[i] ) veloc_x[i] += ( vres[i] - veloc_x[i] ) * 1.00;
	}
}

void FixMuiMeso::gpu_fetch( pair<vector<int>, vector<double> > result ) {
	static int2 grid_cfg;
	static HostScalar<int>    hst_pred(this->lmp,"FixMUI::dev_pred");
	static HostScalar<double> hst_vres(this->lmp,"FixMUI::dev_r");

	if ( !grid_cfg.x )
	{
		grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_scatter_fetch, 0, hipFuncCachePreferL1 );
		hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_scatter_fetch), hipFuncCachePreferL1 );
	}
	if ( hst_pred.n_elem() < atom->nlocal ) {
		hst_pred.grow( atom->nlocal );
		hst_vres.grow( atom->nlocal );
	}

	for(int i=0;i<result.first.size();i++) {
		hst_pred[i] = result.first[i];
		hst_vres[i] = result.second[i];
	}
	gpu_scatter_fetch<<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>>(
		hst_pred,
		hst_vres,
		meso_atom->dev_veloc(0),
		meso_atom->dev_veloc(1),
		meso_atom->dev_veloc(2),
		meso_atom->dev_mask,
		groupbit,
		atom->nlocal );
}

mui::point3d point( double4 x ) {
	mui::point3d p;
	p[0] = x.x;
	p[1] = x.y;
	p[2] = x.z;
	return p;
}

FixMuiMeso::FixMuiMeso(LAMMPS *lmp, int narg, char **arg) :
	Fix(lmp, narg, arg),
	MesoPointers(lmp)
{
	if (narg != 7 && narg != 10) error->all(FLERR,"Illegal fix mui arguments");

	interface = new mui::uniface<mui::default_config>( arg[3] );
	ipush_region = domain->find_region( arg[4] );
	ifetch_region = domain->find_region( arg[5] );
	sample_rc  = atof( arg[6] );

	len_ratio = 1.0;
	vel_ratio = 1.0;
	t_ratio = 1.0;
	multiscale = false;
	if (narg == 10){
		len_ratio = atof( arg[7] );
		vel_ratio = atof( arg[8] );
		t_ratio = atof( arg[9] );
		// tol = atof( arg[10] );
		multiscale = true;
	}
	
}

FixMuiMeso::~FixMuiMeso()
{
	if ( interface ) delete interface;
}

int FixMuiMeso::setmask()
{
	int mask = 0;
	mask |= POST_INTEGRATE;
	mask |= END_OF_STEP;
	return mask;
}

void FixMuiMeso::init()
{
}

void FixMuiMeso::post_integrate()
{
	vector<double4> info = gpu_push();
	int count = info.size();
	if (count == 0) return;

	if (multiscale){
		double vel = 0.0;
		for (int i = 0; i < count; i++) {
			vel += info[i].w;
		}

		vel = vel / count;
		interface->push( "velocity_x", point3d(0.0), vel );
	}
	else{
		for (int i = 0; i < count; i++){
			interface->push( "velocity_x", point(info[i]), info[i].w);
		}
	}

	double time = update->ntimestep * update->dt * t_ratio;
	interface->commit( time );
	interface->barrier( time - 1);
	interface->forget( time - 1 );

}

void FixMuiMeso::end_of_step()
{
	int nlocal = atom->nlocal;

	mui::sampler_shepard_quintic<> quintic(sample_rc * len_ratio);
	mui::temporal_sampler_exact<> texact(tol);

	pair<vector<int>, vector<double4> > pred = gpu_fetch_predicate();
	pair<vector<int>, vector<double> > result;

	double time = update->ntimestep * update->dt * t_ratio;
	double vel = 0.0;

    // fetch average point value
	if (multiscale){
		vel = interface->fetch( "velocity_x", point3d(0.0), time, quintic, texact );
		for (int i = 0; i < nlocal; i++) {
			if ( pred.first[i] ) {
				result.second.push_back( vel/vel_ratio );
			} else
				result.second.push_back( 0.0 );
		}
	}
	else{
		for (int i = 0; i < nlocal; i++) {
			if ( pred.first[i] ) {
				vel = interface->fetch( "velocity_x", point(pred.second[i]), time, quintic, texact );
				result.second.push_back( vel/vel_ratio );
			} else
				result.second.push_back( 0.0 );
		}
	}

	result.first = pred.first;
	gpu_fetch( result );
}

