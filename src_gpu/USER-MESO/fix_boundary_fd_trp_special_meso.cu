#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
     LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
     http://lammps.sandia.gov, Sandia National Laboratories
     Steve Plimpton, sjplimp@sandia.gov

     Copyright (2003) Sandia Corporation.   Under the terms of Contract
     DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
     certain rights in this software.   This software is distributed under
     the GNU General Public License.

     See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

#include "mpi.h"
#include "stdio.h"
#include "string.h"
#include "force.h"
#include "update.h"
#include "error.h"
#include "domain.h"
#include "input.h"
#include "variable.h"

#include "atom_meso.h"
#include "comm_meso.h"
#include "atom_vec_meso.h"
#include "engine_meso.h"
#include "fix_boundary_fd_trp_special_meso.h"
#include "pair_edpd_trp_base_meso.h"

using namespace LAMMPS_NS;
using namespace PNIPAM_COEFFICIENTS;

/* ---------------------------------------------------------------------- */

MesoFixBoundaryFdTRPSpecial::MesoFixBoundaryFdTRPSpecial( LAMMPS *lmp, int narg, char **arg ):
    Fix( lmp, narg, arg ),
    MesoPointers( lmp ),
    wall_type(1),
    cut( 0. ),
    cx( 0. ), cy( 0. ), cz( 0. ),
    ox( 0. ), oy( 0. ), oz( 0. ),
    radius( 0. ), length( 0. ),
    T_H( 1. ), T_C( 1. ),
    A0( 0. ),
    poly( lmp, "MesoFixBoundaryFdTRPSpecial::poly" ),
    pair(NULL)
{
    pair = dynamic_cast<MesoPairEDPDTRPBase*>( force->pair );
    if( !pair ) error->all( FLERR, "<MESO> fix boundary/fc/trp/meso must be used together with pair edpd/pnipam/meso" );

    for( int i = 3; i < narg; i++ ) {
        if( !strcmp( arg[i], "type" ) ) {
            wall_type = atoi( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "cut" ) ) {
            cut = atof( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "T" ) ) {
            T_H = atof( arg[++i] );
            T_C = atof( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "radius" ) ) {
            radius = atof( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "length" ) ) {
            length = atof( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "center" ) ) {
            cx = atof( arg[++i] );
            cy = atof( arg[++i] );
            cz = atof( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "orient" ) ) {
            ox = atof( arg[++i] );
            oy = atof( arg[++i] );
            oz = atof( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "A0" ) ) {
            A0 = atof( arg[++i] );
            continue;
        }
        if( !strcmp( arg[i], "poly" ) ) {
            int order = atoi( arg[++i] );
            poly.grow( order + 1 );
            for( int j = 0; j < order + 1; j++ ) poly[j] = atof( arg[++i] );
            continue;
        }
    }

    if( ( ox == 0. && oy == 0. && oz == 0. ) || radius < 1 || poly.n_elem() == 0 || poly == NULL )
        error->all( FLERR, "Usage: boundary/fc group [type int] [T0 double] [cut double] [radius double] [length double] [center doublex3] [orient doublex3] [poly int doublex?]" );

    double n = std::sqrt( ox * ox + oy * oy + oz * oz );
    ox /= n;
    oy /= n;
    oz /= n;
}

MesoFixBoundaryFdTRPSpecial::~MesoFixBoundaryFdTRPSpecial()
{
}

int MesoFixBoundaryFdTRPSpecial::setmask()
{
    int mask = 0;
    mask |= FixConst::POST_FORCE;
    return mask;
}

void MesoFixBoundaryFdTRPSpecial::init()
{
    if( strcmp( update->integrate_style, "respa" ) == 0 ) {
        fprintf( stderr, "<MESO> RESPA not supported in MesoFixBoundaryFdTRPSpecial. %s %cut\n", __FILE__, __LINE__ );
    }
}

void MesoFixBoundaryFdTRPSpecial::setup( int vflag )
{
    if( strcmp( update->integrate_style, "respa" ) == 0 ) {
        fprintf( stderr, "<MESO> RESPA not supported in MesoFixBoundaryFdTRPSpecial. %s %cut\n", __FILE__, __LINE__ );
    }
    post_force( vflag );
}

__global__ void gpu_fix_boundary_fd_trp_special(
    r64* __restrict coord_x,
    r64* __restrict coord_y,
    r64* __restrict coord_z,
    r64* __restrict veloc_x,
    r64* __restrict veloc_y,
    r64* __restrict veloc_z,
    r64* __restrict force_x,
    r64* __restrict force_y,
    r64* __restrict force_z,
    r64* __restrict T,
    int* __restrict type,
    int* __restrict mask,
    r64* __restrict coefficients,
    const int n_type,
    const int wall_type,
    const int groupbit,
    const int order,
    r64* __restrict poly,
    const r64 A0,
    const r64 T_H,
    const r64 T_C,
    const r64 cx,
    const r64 cy,
    const r64 cz,
    const r64 ox,
    const r64 oy,
    const r64 oz,
    const r64 cut,
    const r64 radius,
    const r64 length,
    const r64 dtinvsqrt,
    const int n )
{
    extern __shared__ r64 coeffs[];
    for( int p = threadIdx.x; p < n_type * n_type * n_coeff; p += blockDim.x )
        coeffs[p] = coefficients[p];
    __syncthreads();

    for( int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x ) {
        if( mask[i] & groupbit ) {
            r64 dx = coord_x[i] - cx;
            r64 dy = coord_y[i] - cy;
            r64 dz = coord_z[i] - cz;
            r64 along = dx * ox + dy * oy + dz * oz;
            r64 perpx = dx - along * ox;
            r64 perpy = dy - along * oy;
            r64 perpz = dz - along * oz;
            r64 d = sqrt( perpx*perpx + perpy*perpy + perpz*perpz );
            if( d > radius - cut ) {
                r64 T0        = ( fmod( along, length ) > ( length * 0.5 ) ) ? T_C : T_H;
                r64 h         = max( min( radius - d, cut ), 0. );
                r64 nx        = perpx / d;
                r64 ny        = perpy / d;
                r64 nz        = perpz / d;
                r64 *coeff_ij = coeffs + ( (type[i]-1) * n_type + (wall_type-1) ) * n_coeff;
                // dissipative force
                r64 gammah    = polyval( h, order, poly );
                gammah   += A0 / h;
                gammah    = max( min( gammah, 1.0 ), 0. );
                gammah   *= coeff_ij[ p_gamma ];
                r64 TT        = 2.0 * T[i] * T0 / ( T[i] + T0 );
                r64 sigmah    = sqrt( 2.0 * TT * gammah );
                r64 v_n_e_n   = veloc_x[i] * nx + veloc_y[i] * ny + veloc_z[i] * nz;
                r64 v_t_x     = veloc_x[i] - v_n_e_n * nx;
                r64 v_t_y     = veloc_y[i] - v_n_e_n * ny;
                r64 v_t_z     = veloc_z[i] - v_n_e_n * nz;
                // balancing random force
                r64 rn        = _SQRT_2 * dtinvsqrt * uniform_TEA<32>( i  , __mantissa( veloc_x[i], veloc_y[i], veloc_z[i] ) );
                // composite force
                r64 v_t       = sqrt( v_t_x*v_t_x + v_t_y*v_t_y + v_t_z*v_t_z );
                if ( v_t > EPSILON ) {
                    r64 e_t_x     = v_t_x / v_t;
                    r64 e_t_y     = v_t_y / v_t;
                    r64 e_t_z     = v_t_z / v_t;
                    force_x[i]   -= gammah * v_t_x + sigmah * rn * e_t_x;
                    force_y[i]   -= gammah * v_t_y + sigmah * rn * e_t_y;
                    force_z[i]   -= gammah * v_t_z + sigmah * rn * e_t_z;
                }
            }
        }
    }
}

void MesoFixBoundaryFdTRPSpecial::post_force( int vflag )
{
    static GridConfig grid_cfg;
    if( !grid_cfg.x ) {
        grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_fix_boundary_fd_trp_special, 0, hipFuncCachePreferL1 );
        hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_fix_boundary_fd_trp_special), hipFuncCachePreferL1 );
    }

    prepare_coeff();

    gpu_fix_boundary_fd_trp_special <<< grid_cfg.x, grid_cfg.y, pair->dev_coefficients.n_byte(), meso_device->stream() >>> (
        meso_atom->dev_coord(0), meso_atom->dev_coord(1), meso_atom->dev_coord(2),
        meso_atom->dev_veloc(0), meso_atom->dev_veloc(1), meso_atom->dev_veloc(2),
        meso_atom->dev_force(0), meso_atom->dev_force(1), meso_atom->dev_force(2),
        meso_atom->dev_T,
        meso_atom->dev_type, meso_atom->dev_mask,
        pair->dev_coefficients,
        atom->ntypes,
        wall_type,
        groupbit,
        poly.n_elem() - 1,
        poly,
        A0,
        T_H, T_C,
        cx, cy, cz,
        ox, oy, oz,
        cut, radius, length,
        std::sqrt( 1./update->dt ),
        atom->nlocal );
}

void MesoFixBoundaryFdTRPSpecial::prepare_coeff() {
    pair->prepare_coeff();
}
