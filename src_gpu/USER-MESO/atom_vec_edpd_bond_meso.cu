#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "domain.h"
#include "modify.h"
#include "fix.h"
#include "memory.h"
#include "error.h"
#include "bond.h"
#include "force.h"

#include "atom_meso.h"
#include "domain_meso.h"
#include "engine_meso.h"
#include "neighbor_meso.h"
#include "atom_vec_edpd_bond_meso.h"

using namespace LAMMPS_NS;

AtomVecEDPDBond::AtomVecEDPDBond( LAMMPS *lmp ) :
    AtomVecDPDBond( lmp ),
    dev_Q( lmp, "AtomVecEDPDBond::dev_q" ),
    dev_T( lmp, "AtomVecEDPDBond::dev_T" ),
    dev_T_pinned( lmp, "AtomVecEDPDBond::dev_T_pinned" ),
    dev_Q_pinned( lmp, "AtomVecEDPDBond::dev_Q_pinned" ),
    dev_therm_merged( lmp, "AtomVecEDPDBond::dev_therm_merged" )
{
    comm_x_only    = 0;
    comm_f_only    = 0;
    mass_type      = 1;
    size_forward   = 4;
    size_border    = 8;
    size_velocity  = 3;
    size_data_atom = 7;
    size_data_vel  = 4;
    xcol_data      = 4;

    cudable        = 1;
    pre_sort     = AtomAttribute::LOCAL  | AtomAttribute::COORD;
    post_sort    = AtomAttribute::LOCAL  | AtomAttribute::ESSENTIAL | AtomAttribute::TEMP | AtomAttribute::MOLE |
                   AtomAttribute::EXCL   | AtomAttribute::BOND;
    pre_border   = AtomAttribute::BORDER | AtomAttribute::ESSENTIAL | AtomAttribute::TEMP | AtomAttribute::MOLE;
    post_border  = AtomAttribute::GHOST  | AtomAttribute::ESSENTIAL | AtomAttribute::TEMP | AtomAttribute::MOLE;
    pre_comm     = AtomAttribute::BORDER | AtomAttribute::COORD     | AtomAttribute::TEMP | AtomAttribute::VELOC;
    post_comm    = AtomAttribute::GHOST  | AtomAttribute::COORD     | AtomAttribute::TEMP | AtomAttribute::VELOC;
    pre_exchange = AtomAttribute::LOCAL  | AtomAttribute::ESSENTIAL | AtomAttribute::TEMP | AtomAttribute::MOLE |
                   AtomAttribute::EXCL   | AtomAttribute::BOND;
    pre_output   = AtomAttribute::LOCAL  | AtomAttribute::ESSENTIAL | AtomAttribute::TEMP | AtomAttribute::FORCE |
                   AtomAttribute::MOLE   | AtomAttribute::EXCL      | AtomAttribute::BOND;

    Q = T = NULL;
}

void AtomVecEDPDBond::copy( int i, int j, int delflag )
{
    tag[j] = tag[i];
    type[j] = type[i];
    mask[j] = mask[i];
    image[j] = image[i];
    x[j][0] = x[i][0];
    x[j][1] = x[i][1];
    x[j][2] = x[i][2];
    v[j][0] = v[i][0];
    v[j][1] = v[i][1];
    v[j][2] = v[i][2];
    T[j] = T[i];

    molecule[j] = molecule[i];

    num_bond[j] = num_bond[i];
    for( int k = 0; k < num_bond[j]; k++ ) {
        bond_type[j][k] = bond_type[i][k];
        bond_atom[j][k] = bond_atom[i][k];
    }

    nspecial[j][0] = nspecial[i][0];
    nspecial[j][1] = nspecial[i][1];
    nspecial[j][2] = nspecial[i][2];
    for( int k = 0; k < nspecial[j][2]; k++ ) special[j][k] = special[i][k];

    if( atom->nextra_grow )
        for( int iextra = 0; iextra < atom->nextra_grow; iextra++ )
            modify->fix[atom->extra_grow[iextra]]->copy_arrays( i, j, delflag );
}

void AtomVecEDPDBond::grow( int n )
{
    unpin_host_array();
    if( n == 0 ) n = max( nmax + growth_inc, ( int )( nmax * growth_mul ) );
    grow_cpu( n );
    grow_device( n );
    pin_host_array();
}

void AtomVecEDPDBond::grow_cpu( int n )
{
    AtomVecDPDBond::grow_cpu( n );

    T = memory->grow( atom->T, nmax, "atom:T" );
    Q = memory->grow( atom->Q, nmax, "atom:Q" );
}

void AtomVecEDPDBond::grow_device( int nmax_new )
{
    AtomVecDPDBond::grow_device( nmax_new );

    // gpu global memory
    meso_atom->dev_Q = dev_Q.grow( nmax_new );
    meso_atom->dev_T = dev_T.grow( nmax_new );
    meso_atom->dev_therm_merged = dev_therm_merged.grow( nmax_new, false, false );

    // texture
    meso_atom->tex_misc("therm").bind( dev_therm_merged );
}

void AtomVecEDPDBond::pin_host_array()
{
    AtomVecDPDBond::pin_host_array();

    if( atom->T ) dev_T_pinned.map_host( atom->nmax, atom->T );
    if( atom->Q ) dev_Q_pinned.map_host( atom->nmax, atom->Q );
}

void AtomVecEDPDBond::unpin_host_array()
{
    AtomVecDPDBond::unpin_host_array();

    dev_T_pinned.unmap_host( atom->T );
    dev_Q_pinned.unmap_host( atom->Q );
}

void AtomVecEDPDBond::transfer_impl(
    std::vector<CUDAEvent> &events, AtomAttribute::Descriptor per_atom_prop, TransferDirection direction,
    int p_beg, int n_atom, int p_stream, int p_inc, int* permute_to, int* permute_from, int action, bool streamed )
{
    AtomVecDPDBond::transfer_impl( events, per_atom_prop, direction, p_beg, n_atom, p_stream, p_inc, permute_to, permute_from, action, streamed );
    p_stream = events.size() + p_inc;

    if( per_atom_prop & AtomAttribute::TEMP ) {
        events.push_back(
            transfer_scalar(
                dev_T_pinned, dev_T, direction, permute_from, p_beg, n_atom, meso_device->stream( p_stream += p_inc ), action ) );
    }
    if( per_atom_prop & AtomAttribute::HEAT ) {
        events.push_back(
            transfer_scalar(
                dev_Q_pinned, dev_Q, direction, permute_from, p_beg, n_atom, meso_device->stream( p_stream += p_inc ), action ) );
    }
}

__global__ void gpu_merge_xvtT_(
    r64* __restrict coord_x, r64* __restrict coord_y, r64* __restrict coord_z,
    r64* __restrict veloc_x, r64* __restrict veloc_y, r64* __restrict veloc_z,
    int* __restrict type, int* __restrict tag,
    r64* __restrict mass, r64* __restrict T,
    float4* __restrict coord_merged,
    float4* __restrict veloc_merged,
    float4* __restrict therm_merged,
    const r64 cx, const r64 cy, const r64 cz,
    const int seed1,
    const int seed2,
    const int p_beg,
    const int p_end )
{
    for( int i  = p_beg + blockDim.x * blockIdx.x + threadIdx.x; i < p_end; i += gridDim.x * blockDim.x ) {
        float4 coord;
        coord.x = coord_x[i] - cx;
        coord.y = coord_y[i] - cy;
        coord.z = coord_z[i] - cz;
        coord.w = __int_as_float( type[i] - 1 );
        coord_merged[i] = coord;

        float4 veloc;
        veloc.x = veloc_x[i];
        veloc.y = veloc_y[i];
        veloc.z = veloc_z[i];
        veloc.w = __uint_as_float( premix_TEA<32>( __brev( tag[i] ), seed1 ) );
        veloc_merged[i] = veloc;

        float4 therm;
        therm.x = T[i];
        therm.y = 1.0 / mass[i];
        therm.w = __uint_as_float( premix_TEA<32>( tag[i], seed2 ) );
        therm_merged[i] = therm;
    }
}


void AtomVecEDPDBond::dp2sp_merged( int seed, int p_beg, int p_end, bool offset )
{
    r64 cx = 0., cy = 0., cz = 0.;
    if( offset ) {
        cx = 0.5 * ( meso_domain->subhi[0] + meso_domain->sublo[0] );
        cy = 0.5 * ( meso_domain->subhi[1] + meso_domain->sublo[1] );
        cz = 0.5 * ( meso_domain->subhi[2] + meso_domain->sublo[2] );
    }

    static GridConfig grid_cfg;
    if( !grid_cfg.x ) {
        grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_merge_xvtT_, 0, hipFuncCachePreferL1 );
        hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_merge_xvtT_), hipFuncCachePreferL1 );
    }

    gpu_merge_xvtT_ <<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>> (
        dev_coord(0), dev_coord(1), dev_coord(2),
        dev_veloc(0), dev_veloc(1), dev_veloc(2),
        dev_type,
        dev_tag,
        dev_mass,
        dev_T,
        dev_coord_merged,
        dev_veloc_merged,
        dev_therm_merged,
        cx, cy, cz,
        seed,
        seed*1664525U+1013904223U,
        p_beg, p_end );
}

/* ---------------------------------------------------------------------- */

int AtomVecEDPDBond::pack_comm( int n, int *list, double *buf,
                                int pbc_flag, int *pbc )
{
    int i, j, m;
    double dx, dy, dz;

    m = 0;
    if( pbc_flag == 0 ) {
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0];
            buf[m++] = x[j][1];
            buf[m++] = x[j][2];
            buf[m++] = T[j];
        }
    } else {
        if( domain->triclinic == 0 ) {
            dx = pbc[0] * domain->xprd;
            dy = pbc[1] * domain->yprd;
            dz = pbc[2] * domain->zprd;
        } else {
            dx = pbc[0] * domain->xprd + pbc[5] * domain->xy + pbc[4] * domain->xz;
            dy = pbc[1] * domain->yprd + pbc[3] * domain->yz;
            dz = pbc[2] * domain->zprd;
        }
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0] + dx;
            buf[m++] = x[j][1] + dy;
            buf[m++] = x[j][2] + dz;
            buf[m++] = T[j];
        }
    }
    return m;
}

int AtomVecEDPDBond::pack_comm_vel( int n, int *list, double *buf,
                                    int pbc_flag, int *pbc )
{
    int i, j, m;
    double dx, dy, dz;

    m = 0;
    if( pbc_flag == 0 ) {
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0];
            buf[m++] = x[j][1];
            buf[m++] = x[j][2];
            buf[m++] = v[j][0];
            buf[m++] = v[j][1];
            buf[m++] = v[j][2];
            buf[m++] = T[j];
        }
    } else {
        if( domain->triclinic == 0 ) {
            dx = pbc[0] * domain->xprd;
            dy = pbc[1] * domain->yprd;
            dz = pbc[2] * domain->zprd;
        } else {
            dx = pbc[0] * domain->xprd + pbc[5] * domain->xy + pbc[4] * domain->xz;
            dy = pbc[1] * domain->yprd + pbc[3] * domain->yz;
            dz = pbc[2] * domain->zprd;
        }
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0] + dx;
            buf[m++] = x[j][1] + dy;
            buf[m++] = x[j][2] + dz;
            buf[m++] = v[j][0];
            buf[m++] = v[j][1];
            buf[m++] = v[j][2];
            buf[m++] = T[j];
        }
    }
    return m;
}

/* ---------------------------------------------------------------------- */

void AtomVecEDPDBond::unpack_comm( int n, int first, double *buf )
{
    int i, m, last;

    m = 0;
    last = first + n;
    for( i = first; i < last; i++ ) {
        x[i][0] = buf[m++];
        x[i][1] = buf[m++];
        x[i][2] = buf[m++];
        T[i]    = buf[m++];
    }
}

void AtomVecEDPDBond::unpack_comm_vel( int n, int first, double *buf )
{
    int i, m, last;

    m = 0;
    last = first + n;
    for( i = first; i < last; i++ ) {
        x[i][0] = buf[m++];
        x[i][1] = buf[m++];
        x[i][2] = buf[m++];
        v[i][0] = buf[m++];
        v[i][1] = buf[m++];
        v[i][2] = buf[m++];
        T[i]    = buf[m++];
    }
}

/* ---------------------------------------------------------------------- */

int AtomVecEDPDBond::pack_border( int n, int *list, double *buf,
                                  int pbc_flag, int *pbc )
{
    int i, j, m;
    double dx, dy, dz;

    m = 0;
    if( pbc_flag == 0 ) {
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0];
            buf[m++] = x[j][1];
            buf[m++] = x[j][2];
            buf[m++] = T[j];
            buf[m++] = tag[j];
            buf[m++] = type[j];
            buf[m++] = mask[j];
            buf[m++] = molecule[j];
        }
    } else {
        if( domain->triclinic == 0 ) {
            dx = pbc[0] * domain->xprd;
            dy = pbc[1] * domain->yprd;
            dz = pbc[2] * domain->zprd;
        } else {
            dx = pbc[0];
            dy = pbc[1];
            dz = pbc[2];
        }
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0] + dx;
            buf[m++] = x[j][1] + dy;
            buf[m++] = x[j][2] + dz;
            buf[m++] = T[j];
            buf[m++] = tag[j];
            buf[m++] = type[j];
            buf[m++] = mask[j];
            buf[m++] = molecule[j];
        }
    }
    return m;
}

int AtomVecEDPDBond::pack_border_vel( int n, int *list, double *buf,
                                      int pbc_flag, int *pbc )
{
    int i, j, m;
    double dx, dy, dz;

    m = 0;
    if( pbc_flag == 0 ) {
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0];
            buf[m++] = x[j][1];
            buf[m++] = x[j][2];
            buf[m++] = v[j][0];
            buf[m++] = v[j][1];
            buf[m++] = v[j][2];
            buf[m++] = T[j];
            buf[m++] = tag[j];
            buf[m++] = type[j];
            buf[m++] = mask[j];
            buf[m++] = molecule[j];
        }
    } else {
        if( domain->triclinic == 0 ) {
            dx = pbc[0] * domain->xprd;
            dy = pbc[1] * domain->yprd;
            dz = pbc[2] * domain->zprd;
        } else {
            dx = pbc[0];
            dy = pbc[1];
            dz = pbc[2];
        }
        for( i = 0; i < n; i++ ) {
            j = list[i];
            buf[m++] = x[j][0] + dx;
            buf[m++] = x[j][1] + dy;
            buf[m++] = x[j][2] + dz;
            buf[m++] = v[j][0];
            buf[m++] = v[j][1];
            buf[m++] = v[j][2];
            buf[m++] = T[j];
            buf[m++] = tag[j];
            buf[m++] = type[j];
            buf[m++] = mask[j];
            buf[m++] = molecule[j];
        }
    }
    return m;
}

/* ---------------------------------------------------------------------- */

void AtomVecEDPDBond::unpack_border( int n, int first, double *buf )
{
    int i, m, last;

    m = 0;
    last = first + n;
    for( i = first; i < last; i++ ) {
        if( i == nmax ) grow( 0 );
        x[i][0] = buf[m++];
        x[i][1] = buf[m++];
        x[i][2] = buf[m++];
        T[i]    = buf[m++];
        tag[i]  = static_cast<int>( buf[m++] );
        type[i] = static_cast<int>( buf[m++] );
        mask[i] = static_cast<int>( buf[m++] );
        molecule[i] = static_cast<int>( buf[m++] );
    }
}

void AtomVecEDPDBond::unpack_border_vel( int n, int first, double *buf )
{
    int i, m, last;

    m = 0;
    last = first + n;
    for( i = first; i < last; i++ ) {
        if( i == nmax ) grow( 0 );
        x[i][0] = buf[m++];
        x[i][1] = buf[m++];
        x[i][2] = buf[m++];
        v[i][0] = buf[m++];
        v[i][1] = buf[m++];
        v[i][2] = buf[m++];
        T[i]    = buf[m++];
        tag[i]  = static_cast<int>( buf[m++] );
        type[i] = static_cast<int>( buf[m++] );
        mask[i] = static_cast<int>( buf[m++] );
        molecule[i] = static_cast<int>( buf[m++] );
    }
}

/* ----------------------------------------------------------------------
     pack data for atom I for sending to another proc
     xyz must be 1st 3 values, so comm::exchange() can test on them
------------------------------------------------------------------------- */

int AtomVecEDPDBond::pack_exchange( int i, double *buf )
{
    int m = 1;
    buf[m++] = x[i][0];
    buf[m++] = x[i][1];
    buf[m++] = x[i][2];
    buf[m++] = v[i][0];
    buf[m++] = v[i][1];
    buf[m++] = v[i][2];
    buf[m++] = T[i];
    buf[m++] = tag[i];
    buf[m++] = type[i];
    buf[m++] = mask[i];
    buf[m++] = image[i];

    buf[m++] = molecule[i];

    buf[m++] = num_bond[i];
    for( int k = 0; k < num_bond[i]; k++ ) {
        buf[m++] = bond_type[i][k];
        buf[m++] = bond_atom[i][k];
    }

    buf[m++] = nspecial[i][0];
    buf[m++] = nspecial[i][1];
    buf[m++] = nspecial[i][2];
    for( int k = 0; k < nspecial[i][2]; k++ ) buf[m++] = special[i][k];

    if( atom->nextra_grow )
        for( int iextra = 0; iextra < atom->nextra_grow; iextra++ )
            m += modify->fix[atom->extra_grow[iextra]]->pack_exchange( i, &buf[m] );

    buf[0] = m;
    return m;
}

/* ---------------------------------------------------------------------- */

int AtomVecEDPDBond::unpack_exchange( double *buf )
{
    int nlocal = atom->nlocal;
    if( nlocal == nmax ) grow( 0 );

    int m = 1;
    x[nlocal][0] = buf[m++];
    x[nlocal][1] = buf[m++];
    x[nlocal][2] = buf[m++];
    v[nlocal][0] = buf[m++];
    v[nlocal][1] = buf[m++];
    v[nlocal][2] = buf[m++];
    T[nlocal]    = buf[m++];
    tag[nlocal]  = static_cast<int>( buf[m++] );
    type[nlocal] = static_cast<int>( buf[m++] );
    mask[nlocal] = static_cast<int>( buf[m++] );
    image[nlocal] = static_cast<int>( buf[m++] );

    molecule[nlocal] = static_cast<int>( buf[m++] );

    num_bond[nlocal] = static_cast<int>( buf[m++] );
    for( int k = 0; k < num_bond[nlocal]; k++ ) {
        bond_type[nlocal][k] = static_cast<int>( buf[m++] );
        bond_atom[nlocal][k] = static_cast<int>( buf[m++] );
    }

    nspecial[nlocal][0] = static_cast<int>( buf[m++] );
    nspecial[nlocal][1] = static_cast<int>( buf[m++] );
    nspecial[nlocal][2] = static_cast<int>( buf[m++] );
    for( int k = 0; k < nspecial[nlocal][2]; k++ )
        special[nlocal][k] = static_cast<int>( buf[m++] );

    if( atom->nextra_grow )
        for( int iextra = 0; iextra < atom->nextra_grow; iextra++ )
            m += modify->fix[atom->extra_grow[iextra]]->
                 unpack_exchange( nlocal, &buf[m] );

    atom->nlocal++;
    return m;
}

/* ----------------------------------------------------------------------
     size of restart data for all atoms owned by this proc
     include extra data stored by fixes
------------------------------------------------------------------------- */

int AtomVecEDPDBond::size_restart()
{
    int n = 0;
    for( int i = 0; i < atom->nlocal; i++ ) n += 14 + 2 * num_bond[i];

    if( atom->nextra_restart )
        for( int iextra = 0; iextra < atom->nextra_restart; iextra++ )
            for( int i = 0; i < atom->nlocal; i++ )
                n += modify->fix[atom->extra_restart[iextra]]->size_restart( i );

    return n;
}

/* ----------------------------------------------------------------------
     pack atom I's data for restart file including extra quantities
     xyz must be 1st 3 values, so that read_restart can test on them
     molecular types may be negative, but write as positive
------------------------------------------------------------------------- */

int AtomVecEDPDBond::pack_restart( int i, double *buf )
{
    int m = 1;
    buf[m++] = x[i][0];
    buf[m++] = x[i][1];
    buf[m++] = x[i][2];
    buf[m++] = v[i][0];
    buf[m++] = v[i][1];
    buf[m++] = v[i][2];
    buf[m++] = T[i];
    buf[m++] = tag[i];
    buf[m++] = type[i];
    buf[m++] = mask[i];
    buf[m++] = image[i];

    buf[m++] = molecule[i];

    buf[m++] = num_bond[i];
    for( int k = 0; k < num_bond[i]; k++ ) {
        buf[m++] = MAX( bond_type[i][k], -bond_type[i][k] );
        buf[m++] = bond_atom[i][k];
    }

    if( atom->nextra_restart )
        for( int iextra = 0; iextra < atom->nextra_restart; iextra++ )
            m += modify->fix[atom->extra_restart[iextra]]->pack_restart( i, &buf[m] );

    buf[0] = m;
    return m;
}

/* ----------------------------------------------------------------------
     unpack data for one atom from restart file including extra quantities
------------------------------------------------------------------------- */

int AtomVecEDPDBond::unpack_restart( double *buf )
{
    int nlocal = atom->nlocal;
    if( nlocal == nmax ) {
        grow( 0 );
        if( atom->nextra_store )
            atom->extra = memory->grow( atom->extra, nmax, atom->nextra_store, "atom:extra" );
    }

    int m = 1;
    x[nlocal][0] = buf[m++];
    x[nlocal][1] = buf[m++];
    x[nlocal][2] = buf[m++];
    v[nlocal][0] = buf[m++];
    v[nlocal][1] = buf[m++];
    v[nlocal][2] = buf[m++];
    T[nlocal]    = buf[m++];
    tag[nlocal]  = static_cast<int>( buf[m++] );
    type[nlocal] = static_cast<int>( buf[m++] );
    mask[nlocal] = static_cast<int>( buf[m++] );
    image[nlocal] = static_cast<int>( buf[m++] );

    molecule[nlocal] = static_cast<int>( buf[m++] );

    num_bond[nlocal] = static_cast<int>( buf[m++] );
    for( int k = 0; k < num_bond[nlocal]; k++ ) {
        bond_type[nlocal][k] = static_cast<int>( buf[m++] );
        bond_atom[nlocal][k] = static_cast<int>( buf[m++] );
    }

    nspecial[nlocal][0] = nspecial[nlocal][1] = nspecial[nlocal][2] = 0;

    double **extra = atom->extra;
    if( atom->nextra_store ) {
        int size = static_cast<int>( buf[0] ) - m;
        for( int i = 0; i < size; i++ ) extra[nlocal][i] = buf[m++];
    }

    atom->nlocal++;
    return m;
}

void AtomVecEDPDBond::data_atom( double *coord, int imagetmp, char **values )
{
    int nlocal = atom->nlocal;
    if( nlocal == nmax ) grow( 0 );

    tag[nlocal] = atoi( values[0] );
    if( tag[nlocal] <= 0 )
        error->one( FLERR, "Invalid atom ID in Atoms section of data file" );

    molecule[nlocal] = atoi( values[1] );

    type[nlocal] = atoi( values[2] );
    if( type[nlocal] <= 0 || type[nlocal] > atom->ntypes )
        error->one( FLERR, "Invalid atom type in Atoms section of data file" );

    x[nlocal][0] = coord[0];
    x[nlocal][1] = coord[1];
    x[nlocal][2] = coord[2];

    T[nlocal]    = atof( values[6] );

    image[nlocal] = imagetmp;

    mask[nlocal] = 1;
    v[nlocal][0] = 0.0;
    v[nlocal][1] = 0.0;
    v[nlocal][2] = 0.0;
    num_bond[nlocal] = 0;

    atom->nlocal++;
}

/* ----------------------------------------------------------------------
     unpack hybrid quantities from one line in Atoms section of data file
     initialize other atom quantities for this sub-style
------------------------------------------------------------------------- */

//int AtomVecEDPDBond::data_atom_hybrid(int nlocal, char **values)
//{
//  v[nlocal][0] = 0.0;
//  v[nlocal][1] = 0.0;
//  v[nlocal][2] = 0.0;
//  T[nlocal]    = 0.0;
//
//  return 0;
//}

/* ----------------------------------------------------------------------
     return # of bytes of allocated memory
------------------------------------------------------------------------- */

bigint AtomVecEDPDBond::memory_usage()
{
    bigint bytes = 0;

    if( atom->memcheck( "tag" ) ) bytes += nmax * sizeof( int );
    if( atom->memcheck( "type" ) ) bytes += nmax * sizeof( int );
    if( atom->memcheck( "mask" ) ) bytes += nmax * sizeof( int );
    if( atom->memcheck( "image" ) ) bytes += nmax * sizeof( int );
    if( atom->memcheck( "x" ) ) bytes += nmax * 3 * sizeof( double );
    if( atom->memcheck( "v" ) ) bytes += nmax * 3 * sizeof( double );
    if( atom->memcheck( "f" ) ) bytes += nmax * 3 * sizeof( double );
    if( atom->memcheck( "i_vf" ) ) bytes += nmax * 6 * sizeof( double );
    if( atom->memcheck( "T" ) ) bytes += nmax * sizeof( double );
    if( atom->memcheck( "Q" ) ) bytes += nmax * sizeof( double );

    return bytes;
}

void AtomVecEDPDBond::force_clear( AtomAttribute::Descriptor range, int vflag )
{
    // clear force on all particles
    // newton flag is always off in MESO-MVV, so never include ghosts
    int p_beg, p_end, n_work;
    resolve_work_range( range, p_beg, p_end );
    if( meso_neighbor->includegroup ) p_end = min( p_end, meso_atom->nfirst );
    n_work = p_end - p_beg;

    dev_force.set( 0.0, meso_device->stream(), p_beg, n_work );
    dev_Q.set( 0.0, meso_device->stream(), p_beg, n_work );
    if( vflag ) dev_virial.set( 0.0, meso_device->stream(), p_beg, n_work );
}

