#include "hip/hip_runtime.h"
#include "mpi.h"
#include "math.h"
#include "stdio.h"
#include "stdlib.h"
#include "atom_vec.h"
#include "update.h"
#include "force.h"
#include "neigh_list.h"
#include "neigh_request.h"
#include "random_mars.h"
#include "memory.h"
#include "error.h"
#include "modify.h"
#include "fix.h"

#include "atom_meso.h"
#include "atom_vec_meso.h"
#include "comm_meso.h"
#include "neighbor_meso.h"
#include "neigh_list_meso.h"
#include "pair_tdpd_meso.h"

using namespace LAMMPS_NS;
using namespace TDPD_COEFFICIENTS;

MesoPairTDPD::MesoPairTDPD( LAMMPS *lmp ) : Pair( lmp ), MesoPointers( lmp ),
    dev_coefficients( lmp, "MesoPairTDPD::dev_coefficients" ),
    n_species( 1 )
{
    split_flag  = 1;
    coeff_ready = false;
    random = NULL;
}

MesoPairTDPD::~MesoPairTDPD()
{
    if( allocated ) {
        memory->destroy( setflag );
        memory->destroy( cut );
        memory->destroy( cutsq );
        memory->destroy( cutinv );
        memory->destroy( a0 );
        memory->destroy( gamma );
        memory->destroy( sigma );
        memory->destroy( s1 );
        memory->destroy( cutc );
        memory->destroy( kappa );
        memory->destroy( s2 );
    }
}

void MesoPairTDPD::allocate()
{
    allocated = 1;
    int n = atom->ntypes;

    memory->create( setflag, n + 1, n + 1, "pair:setflag" );
    memory->create( cutsq,   n + 1, n + 1, "pair:cutsq" );
    memory->create( cut,     n + 1, n + 1, "pair:cut" );
    memory->create( cutinv, n + 1, n + 1, "pair:cutinv" );
    memory->create( a0,      n + 1, n + 1, "pair:a0" );
    memory->create( gamma,   n + 1, n + 1, "pair:gamma" );
    memory->create( sigma,   n + 1, n + 1, "pair:sigma" );
    memory->create( s1,    n + 1, n + 1, "pair:weight_s1" );

    memory->create( cutc,      n + 1, n + 1, n_species, "pair:cutc" );
    memory->create( kappa,   n + 1, n + 1, n_species, "pair:kappa" );
    memory->create( s2,   n + 1, n + 1, n_species, "pair:weight_s2" );

    for( int i = 1; i <= n; i++ )
        for( int j = i; j <= n; j++ )
            setflag[i][j] = 0;

    dev_coefficients.grow( n * n * (n_coeff + n_chemcoeff * n_species) );
}

void MesoPairTDPD::prepare_coeff()
{
    if( coeff_ready ) return;
    if( !allocated ) allocate();

    int n = atom->ntypes;
    static std::vector<r64> coeff_table;
    coeff_table.resize( n * n * (n_coeff + n_chemcoeff * n_species) );
    for( int i = 1; i <= n; i++ ) {
        for( int j = 1; j <= n; j++ ) {
            int cid = ( i - 1 ) * n + ( j - 1 );

            coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_cut   ] = cut[i][j];
            coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_cutsq ] = cutsq[i][j];
            coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_cutinv] = cutinv[i][j];
            coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_s1    ] = s1[i][j];
            coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_a0    ] = a0[i][j];
            coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_gamma ] = gamma[i][j];
            coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_sigma ] = sigma[i][j];

            for( int k = 0; k<n_species; k++) {
                coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_cutc + n_chemcoeff*k  ] = cutc[i][j][k];
                coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_kappa + n_chemcoeff*k ] = kappa[i][j][k];
                coeff_table[ cid * (n_coeff + n_chemcoeff * n_species) + p_s2 + n_chemcoeff*k    ] = s2[i][j][k];
            }

        }
    }
    dev_coefficients.upload( &coeff_table[0], coeff_table.size(), meso_device->stream() );
    coeff_ready = true;
}

template<int evflag>
__global__ void gpu_tdpd(
    texobj tex_coord, texobj tex_veloc,
    r64* __restrict force_x,   r64* __restrict force_y,   r64* __restrict force_z,
    r32** __restrict CONC, r32** __restrict CONF, const uint n_species,
    r64* __restrict virial_xx, r64* __restrict virial_yy, r64* __restrict virial_zz,
    r64* __restrict virial_xy, r64* __restrict virial_xz, r64* __restrict virial_yz,
    int* __restrict pair_count, int* __restrict pair_table,
    r64* __restrict e_pair,
    r64* __restrict coefficients,
    const r64 dt_inv_sqrt,
    const int pair_padding,
    const int n_type,
    const int p_beg,
    const int p_end
)
{
    extern __shared__ r64 coeffs[];
    for( int p = threadIdx.x; p < n_type * n_type * (n_coeff + n_chemcoeff * n_species); p += blockDim.x )
        coeffs[p] = coefficients[p];
    __syncthreads();

    for( int iter = blockIdx.x * blockDim.x + threadIdx.x; ; iter += gridDim.x * blockDim.x ) {
        int i = ( p_beg & WARPALIGN ) + iter;
        if( i >= p_end ) break;
        if( i >= p_beg ) {
            f3u  coord1 = tex1Dfetch<float4>( tex_coord, i );
            f3u  veloc1 = tex1Dfetch<float4>( tex_veloc, i );

            int  n_pair = pair_count[i];
            int *p_pair = pair_table + ( i - __laneid() ) * pair_padding + __laneid();
            r64 fx   = 0., fy   = 0., fz   = 0.;
            r64 vrxx = 0., vryy = 0., vrzz = 0.;
            r64 vrxy = 0., vrxz = 0., vryz = 0.;
            r64 energy = 0.;

            for( int p = 0; p < n_pair; p++ ) {
                int j   = __lds( p_pair );
                p_pair += pair_padding;
                if( ( p & 31 ) == 31 ) p_pair -= 32 * pair_padding - 32;

                f3u coord2   = tex1Dfetch<float4>( tex_coord, j );
                r64 dx       = coord1.x - coord2.x;
                r64 dy       = coord1.y - coord2.y;
                r64 dz       = coord1.z - coord2.z;
                r64 rsq      = dx * dx + dy * dy + dz * dz;
                r64 *coeff_ij = coeffs + ( coord1.i * n_type + coord2.i ) * (n_coeff + n_chemcoeff * n_species);

                // force --------------------------------------------------------------------------
                if( rsq < coeff_ij[p_cutsq] && rsq >= EPSILON_SQ ) {
                    f3u veloc2   = tex1Dfetch<float4>( tex_veloc, j );
                    r64 rn       = gaussian_TEA<4>( veloc1.i > veloc2.i, veloc1.i, veloc2.i );
                    r64 rinv     = rsqrt( rsq );
                    r64 r        = rsq * rinv;
                    r64 dvx      = veloc1.x - veloc2.x;
                    r64 dvy      = veloc1.y - veloc2.y;
                    r64 dvz      = veloc1.z - veloc2.z;
                    r64 dot      = dx * dvx + dy * dvy + dz * dvz;
                    r64 wc       = 1.0 - r * coeff_ij[p_cutinv];
                    r64 wr       = __powd( wc, 0.5 * coeff_ij[p_s1] );
                    // Sigma and Gamma are directly given as parameters.
                    r64 fpair  =  coeff_ij[p_a0] * wc
                                  - ( coeff_ij[p_gamma] * wr * wr * dot * rinv )
                                  + ( coeff_ij[p_sigma] * wr * rn * dt_inv_sqrt );
                    fpair     *= rinv;

                    fx += dx * fpair;
                    fy += dy * fpair;
                    fz += dz * fpair;

                    if( evflag ) {
                        vrxx += dx * dx * fpair;
                        vryy += dy * dy * fpair;
                        vrzz += dz * dz * fpair;
                        vrxy += dx * dy * fpair;
                        vrxz += dx * dz * fpair;
                        vryz += dy * dz * fpair;
                        energy += 0.5 * coeff_ij[p_a0] * coeff_ij[p_cut] * wc * wc;
                    }
                }

                // chemical concentration transport -----------------------------------------------
                for (int k=0; k<n_species; k++) {
                    if ( rsq < (coeff_ij[p_cutc+k*n_chemcoeff] * coeff_ij[p_cutc+k*n_chemcoeff]) && rsq >= EPSILON_SQ ) {
                        r64 rinv    = rsqrt( rsq );
                        r64 r       = rsq * rinv;
                        r64 wcr     = 1.0 - r * __rcp( coeff_ij[p_cutc+k*n_chemcoeff] );
                        r64 wdc     = __powd( wcr, coeff_ij[p_s2+k*n_chemcoeff] );
                        r32 flux    = static_cast<float>(-coeff_ij[p_kappa+k*n_chemcoeff] * wdc) * ( __ldg( CONC[k] + i ) - __ldg( CONC[k] + j ) );
                        CONF [k][i] += flux;        // The flux on the other particle will take care of itself.
                    }
                }
            }

            force_x[i] += fx;
            force_y[i] += fy;
            force_z[i] += fz;
            if( evflag ) {
                e_pair[i] = energy * 0.5;
                virial_xx[i] += vrxx * 0.5;
                virial_yy[i] += vryy * 0.5;
                virial_zz[i] += vrzz * 0.5;
                virial_xy[i] += vrxy * 0.5;
                virial_xz[i] += vrxz * 0.5;
                virial_yz[i] += vryz * 0.5;
            }
        }
    }
}

void MesoPairTDPD::compute_kernel( int eflag, int vflag, int p_beg, int p_end )
{
    if( !coeff_ready ) prepare_coeff();
    MesoNeighList *dlist = meso_neighbor->lists_device[ list->index ];

    int shared_mem_size = atom->ntypes * atom->ntypes * (n_coeff + n_chemcoeff * n_species) * sizeof( r64 );

    if( eflag || vflag ) {
        // evaluate force, energy and virial
        static GridConfig grid_cfg = meso_device->configure_kernel( gpu_tdpd<1>, shared_mem_size );
        gpu_tdpd<1> <<< grid_cfg.x, grid_cfg.y, shared_mem_size, meso_device->stream() >>> (
            meso_atom->tex_coord_merged, meso_atom->tex_veloc_merged,
            meso_atom->dev_force(0), meso_atom->dev_force(1), meso_atom->dev_force(2),
            meso_atom->dev_CONC.ptrs(), meso_atom->dev_CONF.ptrs(), (*(meso_atom->dev_CONC)).d(),
            meso_atom->dev_virial(0), meso_atom->dev_virial(1), meso_atom->dev_virial(2),
            meso_atom->dev_virial(3), meso_atom->dev_virial(4), meso_atom->dev_virial(5),
            dlist->dev_pair_count_core, dlist->dev_pair_table,
            meso_atom->dev_e_pair, dev_coefficients,
            1.0 / sqrt( update->dt ), dlist->n_col,
            atom->ntypes, p_beg, p_end );
    } else {
        // evaluate force only
        static GridConfig grid_cfg = meso_device->configure_kernel( gpu_tdpd<0>, shared_mem_size );
        gpu_tdpd<0> <<< grid_cfg.x, grid_cfg.y, shared_mem_size, meso_device->stream() >>> (
            meso_atom->tex_coord_merged, meso_atom->tex_veloc_merged,
            meso_atom->dev_force(0), meso_atom->dev_force(1), meso_atom->dev_force(2),
            meso_atom->dev_CONC.ptrs(), meso_atom->dev_CONF.ptrs(), (*(meso_atom->dev_CONC)).d(),
            meso_atom->dev_virial(0), meso_atom->dev_virial(1), meso_atom->dev_virial(2),
            meso_atom->dev_virial(3), meso_atom->dev_virial(4), meso_atom->dev_virial(5),
            dlist->dev_pair_count_core, dlist->dev_pair_table,
            meso_atom->dev_e_pair, dev_coefficients,
            1.0 / sqrt( update->dt ), dlist->n_col,
            atom->ntypes, p_beg, p_end );
    }
}

void MesoPairTDPD::compute_bulk( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::BULK, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::LOCAL, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}

void MesoPairTDPD::compute_border( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::BORDER, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::GHOST, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}

void MesoPairTDPD::compute( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::LOCAL, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::ALL, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true );
    compute_kernel( eflag, vflag, p_beg, p_end );
}

uint MesoPairTDPD::seed_now() {
    return premix_TEA<64>( seed, update->ntimestep );
}

void MesoPairTDPD::settings( int narg, char **arg )
{
    if( narg != 3 ) error->all( FLERR, "Illegal pair_style command" );

    cut_global = atof( arg[0] );
    seed = atoi( arg[1] );
    n_species = atoi( arg[2] );
    if( random ) delete random;
    random = new RanMars( lmp, seed % 899999999 + 1 );

    if( allocated ) {
        for( int i = 1; i <= atom->ntypes; i++ )
            for( int j = i + 1; j <= atom->ntypes; j++ )
                if( setflag[i][j] )
                    cut[i][j] = cut_global, cutinv[i][j] = 1.0 / cut_global;
    }
}

void MesoPairTDPD::coeff( int narg, char **arg )
{
    if( narg != 7 + n_chemcoeff*n_species)
        error->all( FLERR, "Incorrect args for pair coefficients" );
    if( !allocated ) allocate();

    int ilo, ihi, jlo, jhi;
    force->bounds( arg[0], atom->ntypes, ilo, ihi );
    force->bounds( arg[1], atom->ntypes, jlo, jhi );

    int p=2;
    double a0_one       = atof( arg[p++] );
    double gamma_one    = atof( arg[p++] );
    double sigma_one    = atof( arg[p++] );
    double s1_one       = atof( arg[p++] );
    double cut_one      = atof( arg[p++] );


    r64 cut_two[n_species], kappa_one[n_species], s2_one[n_species];
    for (int k=0; k<n_species; k++) {
        cut_two[k]     = atof( arg[p++] );
        kappa_one[k]   = atof( arg[p++] );
        s2_one[k]      = atof( arg[p++] );
    }

    int count = 0;
    for( int i = ilo; i <= ihi; i++ ) {
        for( int j = MAX( jlo, i ); j <= jhi; j++ ) {
            a0[i][j]    = a0_one;
            gamma[i][j] = gamma_one;
            sigma[i][j] = sigma_one;
            s1[i][j]    = s1_one;
            cut[i][j]   = cut_one;
            cutsq[i][j] = cut_one * cut_one;
            cutinv[i][j] = 1.0 / cut_one;
            setflag[i][j] = 1;

            // species specific
            for (int k=0; k<n_species; k++) {
                cutc[i][j][k]  = cut_two[k];
                kappa[i][j][k] = kappa_one[k];
                s2[i][j][k]    = s2_one[k];
            }

            count++;
        }
    }

    coeff_ready = false;

    if( count == 0 )
        error->all( FLERR, "Incorrect args for pair coefficients" );
}

/* ----------------------------------------------------------------------
 init specific to this pair style
 ------------------------------------------------------------------------- */

void MesoPairTDPD::init_style()
{
    int i = neighbor->request( this );
    neighbor->requests[i]->cudable = 1;
    neighbor->requests[i]->newton  = 2;
}

/* ----------------------------------------------------------------------
 init for one type pair i,j and corresponding j,i
 ------------------------------------------------------------------------- */

double MesoPairTDPD::init_one( int i, int j )
{
    if( setflag[i][j] == 0 )
        error->all( FLERR, "All pair coeffs are not set" );

    cut[j][i]     = cut[i][j];
    cutinv[j][i]  = cutinv[i][j];
    a0[j][i]      = a0[i][j];
    gamma[j][i]   = gamma[i][j];
    sigma[j][i]   = sigma[i][j];
    s1[j][i]      = s1[i][j];
    for (int k=0; k<n_species; k++) {
        cutc[j][i][k]    = cutc[i][j][k];
        kappa[j][i][k]   = kappa[i][j][k];
        s2[j][i][k]      = s2[i][j][k];
    }

    return cut[i][j];
}

/* ----------------------------------------------------------------------
 proc 0 writes to restart file
 ------------------------------------------------------------------------- */

void MesoPairTDPD::write_restart( FILE *fp )
{
    write_restart_settings( fp );

    for( int i = 1; i <= atom->ntypes; i++ ) {
        for( int j = i; j <= atom->ntypes; j++ ) {
            fwrite( &setflag[i][j], sizeof( int ), 1, fp );
            if( setflag[i][j] ) {
                fwrite( &a0[i][j], sizeof( double ), 1, fp );
                fwrite( &gamma[i][j], sizeof( double ), 1, fp );
                fwrite( &sigma[i][j], sizeof( double ), 1, fp );
                fwrite( &s1[i][j], sizeof( double ), 1, fp );
                fwrite( &cut[i][j], sizeof( double ), 1, fp );
                for (int k=0; k<n_species; k++) {
                    fwrite( &cutc[i][j][k], sizeof( double ), 1, fp );
                    fwrite( &kappa[i][j][k], sizeof( double ), 1, fp );
                    fwrite( &s2[i][j][k], sizeof( double ), 1, fp );
                }
            }
        }
    }
}

/* ----------------------------------------------------------------------
 proc 0 reads from restart file, bcasts
 ------------------------------------------------------------------------- */

void MesoPairTDPD::read_restart( FILE *fp )
{
    read_restart_settings( fp );

    allocate();

    int i, j;
    int me = comm->me;
    for( i = 1; i <= atom->ntypes; i++ ) {
        for( j = i; j <= atom->ntypes; j++ ) {
            if( me == 0 )
                fread( &setflag[i][j], sizeof( int ), 1, fp );
            MPI_Bcast( &setflag[i][j], 1, MPI_INT, 0, world );
            if( setflag[i][j] ) {
                if( me == 0 ) {
                    fread( &a0[i][j], sizeof( double ), 1, fp );
                    fread( &gamma[i][j], sizeof( double ), 1, fp );
                    fread( &sigma[i][j], sizeof( double ), 1, fp );
                    fread( &s1[i][j], sizeof( double ), 1, fp );
                    fread( &cut[i][j], sizeof( double ), 1, fp );
                    for (int k=0; k<n_species; k++) {
                        fread( &cutc[i][j][k], sizeof( double ), 1, fp );
                        fread( &kappa[i][j][k], sizeof( double ), 1, fp );
                        fread( &s2[i][j][k], sizeof( double ), 1, fp );
                    }
                }
                MPI_Bcast( &a0[i][j], 1, MPI_DOUBLE, 0, world );
                MPI_Bcast( &gamma[i][j], 1, MPI_DOUBLE, 0, world );
                MPI_Bcast( &sigma[i][j], 1, MPI_DOUBLE, 0, world );
                MPI_Bcast( &s1[i][j], 1, MPI_DOUBLE, 0, world );
                MPI_Bcast( &cut[i][j], 1, MPI_DOUBLE, 0, world );
                for (int k=0; k<n_species; k++) {
                    MPI_Bcast( &cutc[i][j][k], 1, MPI_DOUBLE, 0, world );
                    MPI_Bcast( &kappa[i][j][k], 1, MPI_DOUBLE, 0, world );
                    MPI_Bcast( &s2[i][j][k], 1, MPI_DOUBLE, 0, world );
                }
                cutinv[i][j] = 1.0 / cut[i][j];
            }
        }
    }
}

/* ----------------------------------------------------------------------
 proc 0 writes to restart file
 ------------------------------------------------------------------------- */

void MesoPairTDPD::write_restart_settings( FILE *fp )
{
    fwrite( &cut_global, sizeof( double ), 1, fp );
    fwrite( &seed, sizeof( int ), 1, fp );
    fwrite( &mix_flag, sizeof( int ), 1, fp );
}

/* ----------------------------------------------------------------------
 proc 0 reads from restart file, bcasts
 ------------------------------------------------------------------------- */

void MesoPairTDPD::read_restart_settings( FILE *fp )
{
    if( comm->me == 0 ) {
        fread( &cut_global, sizeof( double ), 1, fp );
        fread( &seed, sizeof( int ), 1, fp );
        fread( &mix_flag, sizeof( int ), 1, fp );
    }
    MPI_Bcast( &cut_global, 1, MPI_DOUBLE, 0, world );
    MPI_Bcast( &seed, 1, MPI_INT, 0, world );
    MPI_Bcast( &mix_flag, 1, MPI_INT, 0, world );

    if( random ) delete random;
    random = new RanMars( lmp, seed % 899999999 + 1 );
}

/* ---------------------------------------------------------------------- */

double MesoPairTDPD::single( int i, int j, int itype, int jtype, double rsq,
                             double factor_coul, double factor_dpd, double &fforce )
{
    double r, rinv, wr, phi;

    r = sqrt( rsq );
    if( r < EPSILON ) {
        fforce = 0.0;
        return 0.5 * a0[itype][jtype] * cut[itype][jtype];
    }

    rinv = 1.0 / r;

    wr = 1.0 - r * cutinv[itype][jtype];
    fforce = a0[itype][jtype] * wr * factor_dpd * rinv;

    phi = 0.5 * a0[itype][jtype] * cut[itype][jtype] * wr * wr;
    return factor_dpd * phi;
}

