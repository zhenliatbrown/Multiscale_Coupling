#include "hip/hip_runtime.h"
#include "mpi.h"
#include "math.h"
#include "stdio.h"
#include "stdlib.h"
#include "atom_vec.h"
#include "update.h"
#include "force.h"
#include "neigh_list.h"
#include "neigh_request.h"
#include "random_mars.h"
#include "memory.h"
#include "error.h"
#include "modify.h"
#include "fix.h"

#include "atom_meso.h"
#include "atom_vec_meso.h"
#include "comm_meso.h"
#include "neighbor_meso.h"
#include "neigh_list_meso.h"
#include "pair_dpd_fast_meso.h"

using namespace LAMMPS_NS;
using namespace DPD_COEFFICIENTS;

MesoPairDPDFast::MesoPairDPDFast( LAMMPS *lmp ) : Pair( lmp ), MesoPointers( lmp ),
    dev_coefficients( lmp, "MesoPairDPDFast::dev_coefficients" )
{
    split_flag  = 1;
    coeff_ready = false;
    random = NULL;
}

MesoPairDPDFast::~MesoPairDPDFast()
{
    if( allocated ) {
        memory->destroy( setflag );
        memory->destroy( cutsq );
        memory->destroy( cut );
        memory->destroy( cut_inv );
        memory->destroy( a0 );
        memory->destroy( gamma );
        memory->destroy( sigma );
        memory->destroy( expw );
    }
}

void MesoPairDPDFast::allocate()
{
    allocated = 1;
    int n = atom->ntypes;

    memory->create( setflag, n + 1, n + 1, "pair:setflag" );
    memory->create( cutsq,   n + 1, n + 1, "pair:cutsq" );
    memory->create( cut,     n + 1, n + 1, "pair:cut" );
    memory->create( cut_inv, n + 1, n + 1, "pair:cut_inv" );
    memory->create( a0,      n + 1, n + 1, "pair:a0" );
    memory->create( gamma,   n + 1, n + 1, "pair:gamma" );
    memory->create( sigma,   n + 1, n + 1, "pair:sigma" );
    memory->create( expw,    n + 1, n + 1, "pair:weight_exp" );
    for( int i = 1; i <= n; i++ )
        for( int j = i; j <= n; j++ )
            setflag[i][j] = 0;

    dev_coefficients.grow( n * n * n_coeff );
}

void MesoPairDPDFast::prepare_coeff()
{
    if( coeff_ready ) return;
    if( !allocated ) allocate();

    int n = atom->ntypes;
    coeff_table.resize( n * n * n_coeff );
    for( int i = 1; i <= n; i++ ) {
        for( int j = 1; j <= n; j++ ) {
            int cid = ( i - 1 ) * n + ( j - 1 );
            coeff_table[ cid * n_coeff + p_cut   ] = cut[i][j];
            coeff_table[ cid * n_coeff + p_cutsq ] = cutsq[i][j];
            coeff_table[ cid * n_coeff + p_cutinv] = cut_inv[i][j];
            coeff_table[ cid * n_coeff + p_expw  ] = expw[i][j];
            coeff_table[ cid * n_coeff + p_a0    ] = a0[i][j];
            coeff_table[ cid * n_coeff + p_gamma ] = gamma[i][j];
            coeff_table[ cid * n_coeff + p_sigma ] = sigma[i][j];
        }
    }
    dev_coefficients.upload( &coeff_table[0], coeff_table.size(), meso_device->stream() );
    coeff_ready = true;
}

template<int evflag>
__global__ void gpu_dpd_fast(
    texobj tex_coord, texobj tex_veloc,
    r64* __restrict force_x,   r64* __restrict force_y,   r64* __restrict force_z,
    r64* __restrict virial_xx, r64* __restrict virial_yy, r64* __restrict virial_zz,
    r64* __restrict virial_xy, r64* __restrict virial_xz, r64* __restrict virial_yz,
    int* __restrict pair_count, int* __restrict pair_table,
    r64* __restrict e_pair,
    r32* __restrict coefficients,
    const r32 dt_inv_sqrt,
    const int pair_padding,
    const int n_type,
    const int p_beg,
    const int p_end,
    const int n_part )
{
    int block_per_part = gridDim.x / n_part;
    int part_id = blockIdx.x / block_per_part;
    if( part_id >= n_part ) return;
    int part_size = block_per_part * blockDim.x;
    int id_in_partition = blockIdx.x % block_per_part * blockDim.x + threadIdx.x;

    extern __shared__ r32 coeffs[];
    for( int p = threadIdx.x; p < n_type * n_type * n_coeff; p += blockDim.x )
        coeffs[p] = coefficients[p];
    __syncthreads();

    for( int iter = id_in_partition; ; iter += part_size ) {
        int i = ( p_beg & WARPALIGN ) + iter;
        if( i >= p_end ) break;
        if( i >= p_beg ) {
            f3u  coord1 = tex1Dfetch<float4>( tex_coord, i );
            f3u  veloc1 = tex1Dfetch<float4>( tex_veloc,  i );
            int  n_pair = pair_count[i];
            int *p_pair = pair_table + ( i - __laneid() + part_id ) * pair_padding + __laneid();
            r32 fx   = 0.f, fy   = 0.f, fz   = 0.f;
            r32 vrxx = 0.f, vryy = 0.f, vrzz = 0.f;
            r32 vrxy = 0.f, vrxz = 0.f, vryz = 0.f;
            r32 energy = 0.f;

            for( int p = part_id; p < n_pair; p += n_part ) {
                int j   = __lds( p_pair );
                p_pair += pair_padding * n_part;
                if( ( p & 31 ) + n_part >= WARPSZ ) p_pair -= WARPSZ * pair_padding - WARPSZ;

                f3u coord2   = tex1Dfetch<float4>( tex_coord, j );
                r32 dx       = coord1.x - coord2.x;
                r32 dy       = coord1.y - coord2.y;
                r32 dz       = coord1.z - coord2.z;
                r32 rsq      = dx * dx + dy * dy + dz * dz;
                r32 *coeff_ij = coeffs + ( coord1.i * n_type + coord2.i ) * n_coeff;

                if( rsq < coeff_ij[p_cutsq] && rsq >= EPSILON_SQ ) {
                    f3u veloc2   = tex1Dfetch<float4>( tex_veloc, j );
                    r32 rn       = gaussian_TEA_fast<4>( veloc1.i > veloc2.i, veloc1.i, veloc2.i );
                    r32 rinv     = rsqrtf( rsq );
                    r32 r        = rsq * rinv;
                    r32 dvx      = veloc1.x - veloc2.x;
                    r32 dvy      = veloc1.y - veloc2.y;
                    r32 dvz      = veloc1.z - veloc2.z;
                    r32 dot      = dx * dvx + dy * dvy + dz * dvz;
                    r32 wc       = 1.0f - r * coeff_ij[p_cutinv];
                    r32 wr       = powf( wc, coeff_ij[p_expw] );

                    r32 fpair  =  coeff_ij[p_a0] * wc
                                  - ( coeff_ij[p_gamma] * wr * wr * dot * rinv )
                                  + ( coeff_ij[p_sigma] * wr * rn * dt_inv_sqrt );
                    fpair     *= rinv;

                    fx += dx * fpair;
                    fy += dy * fpair;
                    fz += dz * fpair;

                    if( evflag ) {
                        vrxx += dx * dx * fpair;
                        vryy += dy * dy * fpair;
                        vrzz += dz * dz * fpair;
                        vrxy += dx * dy * fpair;
                        vrxz += dx * dz * fpair;
                        vryz += dy * dz * fpair;
                        energy += 0.5f * coeff_ij[p_a0] * coeff_ij[p_cut] * wc * wc;
                    }
                }
            }

            if( n_part == 1 ) {
                force_x[i] += fx;
                force_y[i] += fy;
                force_z[i] += fz;
                if( evflag ) {
                    virial_xx[i] += vrxx * 0.5f;
                    virial_yy[i] += vryy * 0.5f;
                    virial_zz[i] += vrzz * 0.5f;
                    virial_xy[i] += vrxy * 0.5f;
                    virial_xz[i] += vrxz * 0.5f;
                    virial_yz[i] += vryz * 0.5f;
                    e_pair[i] = energy * 0.5f;
                }
            } else {
                atomic_add( force_x + i, fx );
                atomic_add( force_y + i, fy );
                atomic_add( force_z + i, fz );
                if( evflag ) {
                    atomic_add( virial_xx + i, vrxx * 0.5f );
                    atomic_add( virial_yy + i, vryy * 0.5f );
                    atomic_add( virial_zz + i, vrzz * 0.5f );
                    atomic_add( virial_xy + i, vrxy * 0.5f );
                    atomic_add( virial_yz + i, vryz * 0.5f );
                    atomic_add( virial_xz + i, vrxz * 0.5f );
                    atomic_add( e_pair + i, energy * 0.5f );
                }
            }
        }
    }
}

void MesoPairDPDFast::compute_kernel( int eflag, int vflag, int p_beg, int p_end )
{
    if( !coeff_ready ) prepare_coeff();
    MesoNeighList *dlist = meso_neighbor->lists_device[ list->index ];

    int shared_mem_size = atom->ntypes * atom->ntypes * n_coeff * sizeof( r32 );

    if( eflag || vflag ) {
        // evaluate force, energy and virial
        static GridConfig grid_cfg = meso_device->configure_kernel( gpu_dpd_fast<1>, shared_mem_size );
        gpu_dpd_fast<1> <<< grid_cfg.x, grid_cfg.y, shared_mem_size, meso_device->stream() >>> (
            meso_atom->tex_coord_merged, meso_atom->tex_veloc_merged,
            meso_atom->dev_force(0),   meso_atom->dev_force(1),   meso_atom->dev_force(2),
            meso_atom->dev_virial(0), meso_atom->dev_virial(1), meso_atom->dev_virial(2),
            meso_atom->dev_virial(3), meso_atom->dev_virial(4), meso_atom->dev_virial(5),
            dlist->dev_pair_count_core, dlist->dev_pair_table,
            meso_atom->dev_e_pair, dev_coefficients,
            1.0 / sqrt( update->dt ), dlist->n_col,
            atom->ntypes, p_beg, p_end, grid_cfg.partition( p_end - p_beg, WARPSZ ) );
    } else {
        // evaluate force only
        static GridConfig grid_cfg = meso_device->configure_kernel( gpu_dpd_fast<0>, shared_mem_size );
        gpu_dpd_fast<0> <<< grid_cfg.x, grid_cfg.y, shared_mem_size, meso_device->stream() >>> (
            meso_atom->tex_coord_merged, meso_atom->tex_veloc_merged,
            meso_atom->dev_force(0),   meso_atom->dev_force(1),   meso_atom->dev_force(2),
            meso_atom->dev_virial(0), meso_atom->dev_virial(1), meso_atom->dev_virial(2),
            meso_atom->dev_virial(3), meso_atom->dev_virial(4), meso_atom->dev_virial(5),
            dlist->dev_pair_count_core, dlist->dev_pair_table,
            meso_atom->dev_e_pair, dev_coefficients,
            1.0 / sqrt( update->dt ), dlist->n_col,
            atom->ntypes, p_beg, p_end, grid_cfg.partition( p_end - p_beg, WARPSZ ) );
    }
}

void MesoPairDPDFast::compute_bulk( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::BULK, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::LOCAL, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}

void MesoPairDPDFast::compute_border( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::BORDER, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::GHOST, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}

void MesoPairDPDFast::compute( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::LOCAL, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::ALL, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}

uint MesoPairDPDFast::seed_now() {
    return premix_TEA<64>( seed, update->ntimestep );
}

void MesoPairDPDFast::settings( int narg, char **arg )
{
    if( narg != 2 ) error->all( FLERR, "Illegal pair_style command" );

    cut_global = atof( arg[0] );
    seed = atoi( arg[1] );
    if( random ) delete random;
    random = new RanMars( lmp, seed % 899999999 + 1 );

    // reset cutoffs that have been explicitly set
    if( allocated ) {
        for( int i = 1; i <= atom->ntypes; i++ )
            for( int j = i + 1; j <= atom->ntypes; j++ )
                if( setflag[i][j] )
                    cut[i][j] = cut_global, cut_inv[i][j] = 1.0 / cut_global;
    }
}

void MesoPairDPDFast::coeff( int narg, char **arg )
{
    if( narg < 6 || narg > 7 )
        error->all( FLERR, "Incorrect args for pair coefficients" );
    if( !allocated )
        allocate();

    int ilo, ihi, jlo, jhi;
    force->bounds( arg[0], atom->ntypes, ilo, ihi );
    force->bounds( arg[1], atom->ntypes, jlo, jhi );

    float a0_one = atof( arg[2] );
    float gamma_one = atof( arg[3] );
    float sigma_one = atof( arg[4] );
    float expw_one = atof( arg[5] );
    float cut_one = cut_global;
    if( narg == 7 ) cut_one = atof( arg[6] );

    int count = 0;
    for( int i = ilo; i <= ihi; i++ ) {
        for( int j = MAX( jlo, i ); j <= jhi; j++ ) {
            a0[i][j] = a0_one;
            gamma[i][j] = gamma_one;
            sigma[i][j] = sigma_one;
            expw[i][j] = expw_one;
            cut[i][j] = cut_one;
            cutsq[i][j] = cut_one * cut_one;
            cut_inv[i][j] = 1.0 / cut_one;
            setflag[i][j] = 1;
            count++;
        }
    }

    coeff_ready = false;

    if( count == 0 )
        error->all( FLERR, "Incorrect args for pair coefficients" );
}

/* ----------------------------------------------------------------------
 init specific to this pair style
 ------------------------------------------------------------------------- */

void MesoPairDPDFast::init_style()
{
    int i = neighbor->request( this );
    neighbor->requests[i]->cudable = 1;
    neighbor->requests[i]->newton  = 2;
}

/* ----------------------------------------------------------------------
 init for one type pair i,j and corresponding j,i
 ------------------------------------------------------------------------- */

double MesoPairDPDFast::init_one( int i, int j )
{
    if( setflag[i][j] == 0 )
        error->all( FLERR, "All pair coeffs are not set" );

    cut[j][i]     = cut[i][j];
    cut_inv[j][i] = cut_inv[i][j];
    a0[j][i]      = a0[i][j];
    gamma[j][i]   = gamma[i][j];
    sigma[j][i]   = sigma[i][j];
    expw[j][i] = expw[i][j];

    return cut[i][j];
}

/* ----------------------------------------------------------------------
 proc 0 writes to restart file
 ------------------------------------------------------------------------- */

void MesoPairDPDFast::write_restart( FILE *fp )
{
    write_restart_settings( fp );

    for( int i = 1; i <= atom->ntypes; i++ ) {
        for( int j = i; j <= atom->ntypes; j++ ) {
            fwrite( &setflag[i][j], sizeof( int ), 1, fp );
            if( setflag[i][j] ) {
                fwrite( &a0[i][j], sizeof( float ), 1, fp );
                fwrite( &gamma[i][j], sizeof( float ), 1, fp );
                fwrite( &sigma[i][j], sizeof( float ), 1, fp );
                fwrite( &expw[i][j], sizeof( float ), 1, fp );
                fwrite( &cut[i][j], sizeof( float ), 1, fp );
            }
        }
    }
}

/* ----------------------------------------------------------------------
 proc 0 reads from restart file, bcasts
 ------------------------------------------------------------------------- */

void MesoPairDPDFast::read_restart( FILE *fp )
{
    read_restart_settings( fp );

    allocate();

    int i, j;
    int me = comm->me;
    for( i = 1; i <= atom->ntypes; i++ ) {
        for( j = i; j <= atom->ntypes; j++ ) {
            if( me == 0 )
                fread( &setflag[i][j], sizeof( int ), 1, fp );
            MPI_Bcast( &setflag[i][j], 1, MPI_INT, 0, world );
            if( setflag[i][j] ) {
                if( me == 0 ) {
                    fread( &a0[i][j], sizeof( float ), 1, fp );
                    fread( &gamma[i][j], sizeof( float ), 1, fp );
                    fread( &sigma[i][j], sizeof( float ), 1, fp );
                    fread( &expw[i][j], sizeof( float ), 1, fp );
                    fread( &cut[i][j], sizeof( float ), 1, fp );
                }
                MPI_Bcast( &a0[i][j], 1, MPI_FLOAT, 0, world );
                MPI_Bcast( &gamma[i][j], 1, MPI_FLOAT, 0, world );
                MPI_Bcast( &sigma[i][j], 1, MPI_FLOAT, 0, world );
                MPI_Bcast( &expw[i][j], 1, MPI_FLOAT, 0, world );
                MPI_Bcast( &cut[i][j], 1, MPI_FLOAT, 0, world );
                cut_inv[i][j] = 1.0 / cut[i][j];
            }
        }
    }
}

/* ----------------------------------------------------------------------
 proc 0 writes to restart file
 ------------------------------------------------------------------------- */

void MesoPairDPDFast::write_restart_settings( FILE *fp )
{
    fwrite( &cut_global, sizeof( float ), 1, fp );
    fwrite( &seed, sizeof( int ), 1, fp );
    fwrite( &mix_flag, sizeof( int ), 1, fp );
}

/* ----------------------------------------------------------------------
 proc 0 reads from restart file, bcasts
 ------------------------------------------------------------------------- */

void MesoPairDPDFast::read_restart_settings( FILE *fp )
{
    if( comm->me == 0 ) {
        fread( &cut_global, sizeof( float ), 1, fp );
        fread( &seed, sizeof( int ), 1, fp );
        fread( &mix_flag, sizeof( int ), 1, fp );
    }
    MPI_Bcast( &cut_global, 1, MPI_FLOAT, 0, world );
    MPI_Bcast( &seed, 1, MPI_INT, 0, world );
    MPI_Bcast( &mix_flag, 1, MPI_INT, 0, world );

    if( random ) delete random;
    random = new RanMars( lmp, seed % 899999999 + 1 );
}
