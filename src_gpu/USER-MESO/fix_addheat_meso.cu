#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
     LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
     http://lammps.sandia.gov, Sandia National Laboratories
     Steve Plimpton, sjplimp@sandia.gov

     Copyright (2003) Sandia Corporation.   Under the terms of Contract
     DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
     certain rights in this software.   This software is distributed under
     the GNU General Public License.

     See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* Exerting force to induce the periodic Poiseuille flow
 * Adapted from the CPU version fix_zl_force first written by:
 * Zhen Li, Crunch Group, Division of Applied Mathematics, Brown University
 * June, 2013
 */

#include "mpi.h"
#include "stdio.h"
#include "string.h"
#include "force.h"
#include "update.h"
#include "error.h"
#include "domain.h"

#include "atom_meso.h"
#include "comm_meso.h"
#include "atom_vec_meso.h"
#include "engine_meso.h"
#include "fix_addheat_meso.h"

using namespace LAMMPS_NS;

/* ---------------------------------------------------------------------- */

MesoFixAddHeat::MesoFixAddHeat( LAMMPS *lmp, int narg, char **arg ):
    Fix( lmp, narg, arg ),
    MesoPointers( lmp )
{
    if( narg < 4 ) error->all( FLERR, "Illegal fix CUDAPoiseuille command" );

    int parg = 3;
    heat  = atof( arg[parg++] );
}

int MesoFixAddHeat::setmask()
{
    int mask = 0;
    mask |= FixConst::POST_FORCE;
    return mask;
}

void MesoFixAddHeat::init()
{
    if( strcmp( update->integrate_style, "respa" ) == 0 ) {
        fprintf( stderr, "<MESO> RESPA not supported in MesoFixAddHeat. %s %d\n", __FILE__, __LINE__ );
    }
}

void MesoFixAddHeat::setup( int vflag )
{
    if( strcmp( update->integrate_style, "respa" ) == 0 ) {
        fprintf( stderr, "<MESO> RESPA not supported in MesoFixAddHeat. %s %d\n", __FILE__, __LINE__ );
    }
    post_force( vflag );
}

__global__ void gpu_fix_add_heat(
    r64* __restrict Q,
    int* __restrict mask,
    const r64 heat,
    const int groupbit,
    const int n )
{
    for( int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x ) {
        if( mask[i] & groupbit ) Q[i] += heat;
    }
}

void MesoFixAddHeat::post_force( int vflag )
{
    static GridConfig grid_cfg;
    if( !grid_cfg.x ) {
        grid_cfg = meso_device->occu_calc.right_peak( 0, gpu_fix_add_heat, 0, hipFuncCachePreferL1 );
        hipFuncSetCacheConfig(reinterpret_cast<const void*>( gpu_fix_add_heat), hipFuncCachePreferL1 );
    }

    gpu_fix_add_heat <<< grid_cfg.x, grid_cfg.y, 0, meso_device->stream() >>> (
        meso_atom->dev_Q,
        meso_atom->dev_mask,
        heat,
        groupbit,
        atom->nlocal );
}
