#include "hip/hip_runtime.h"
#include "pair_md_meso.h"

#include "atom.h"
#include "error.h"
#include "force.h"
#include "math.h"
#include "math_const.h"
#include "memory.h"
#include "mpi.h"
#include "neigh_list.h"
#include "neigh_request.h"
#include "stdio.h"
#include "stdlib.h"
#include "update.h"

#include "atom_meso.h"
#include "atom_vec_meso.h"
#include "comm_meso.h"
#include "neighbor_meso.h"
#include "neigh_list_meso.h"

using namespace LAMMPS_NS;
using namespace MD_COEFFICIENTS;
using namespace MathConst;

MesoPairMD::MesoPairMD( LAMMPS *lmp) : Pair( lmp ), MesoPointers( lmp ),
    dev_coefficients( lmp, "MesoPairMD::dev_coefficients" )
{
    split_flag = 0;
    coeff_ready = false;
}

MesoPairMD::~MesoPairMD()
{
    if ( !allocated ) return;

    memory->destroy(setflag);
    
    memory->destroy(cut);
    memory->destroy(cutsq);
    memory->destroy(epsilon);
    memory->destroy(sigma);
    memory->destroy(lj1);
    memory->destroy(lj2);
    memory->destroy(lj3);
    memory->destroy(lj4);
    memory->destroy(offset);
}

void MesoPairMD::allocate()
{
    allocated = 1;
    int n = atom->ntypes;

    memory->create( setflag, n + 1, n + 1, "pair::setflag");

    for (int i = 1; i <= n; i++)
        for (int j = 1; j <= n; j++)
            setflag[i][j];

    memory->create( cut, n + 1, n + 1, "pair::cut");
    memory->create( cutsq, n + 1, n + 1, "pair::cutsq");

    memory->create( epsilon, n + 1, n + 1, "pair::epsilon");
    memory->create( sigma, n + 1, n + 1, "pair::sigma");
    memory->create( lj1, n + 1, n + 1, "pair::lj1");
    memory->create( lj2, n + 1, n + 1, "pair::lj2");
    memory->create( lj3, n + 1, n + 1, "pair::lj3");
    memory->create( lj4, n + 1, n + 1, "pair::lj4");
    memory->create( offset, n + 1, n + 1, "pair::offset");

    dev_coefficients.grow( n * n * n_coeff );
}

void MesoPairMD::prepare_coeff()
{
    if (coeff_ready) return;
    if (!allocated) allocate();

    int n = atom->ntypes;
    coeff_table.resize( n * n * n_coeff);

    for ( int i = 1; i <= n; i++ )
    {
        for ( int j = 1; j <= n; j++)
        {
            int cid = (i - 1) * n + (j - 1);
            coeff_table[ cid * n_coeff + p_cut ] = cut[i][j];
            coeff_table[ cid * n_coeff + p_cutsq] = cutsq[i][j];
            coeff_table[ cid * n_coeff + p_epsilon] = epsilon[i][j];
            coeff_table[ cid * n_coeff + p_sigma] = sigma[i][j];
            coeff_table[ cid * n_coeff + p_lj1] = lj1[i][j];
            coeff_table[ cid * n_coeff + p_lj2] = lj2[i][j];
            coeff_table[ cid * n_coeff + p_lj3] = lj3[i][j];
            coeff_table[ cid * n_coeff + p_lj4] = lj4[i][j];
            coeff_table[ cid * n_coeff + p_offset] = offset[i][j];
        }
    }

    dev_coefficients.upload( &coeff_table[0], coeff_table.size(), meso_device->stream() );
    coeff_ready = true;
}

template<int evflag>
__global__ void gpu_pair_md(
    texobj tex_coord, texobj tex_veloc,
    r64* __restrict force_x,   r64* __restrict force_y,   r64* __restrict force_z,
    r64* __restrict virial_xx, r64* __restrict virial_yy, r64* __restrict virial_zz,
    r64* __restrict virial_xy, r64* __restrict virial_xz, r64* __restrict virial_yz,
    int* __restrict pair_count, int* __restrict pair_table,
    r64* __restrict e_pair, r64* __restrict coefficients,
    const int pair_padding,
    const int n_type,
    const int p_beg,
    const int p_end,
    const int n_part)
{
    int block_per_part = gridDim.x / n_part;
    int part_id = blockIdx.x / block_per_part;
    if ( part_id >= n_part ) return;
    int part_size = block_per_part * blockDim.x;
    int id_in_partition = blockIdx.x % block_per_part * blockDim.x + threadIdx.x;
    
    extern __shared__ r64 coeffs[];
    for ( int p = threadIdx.x; p < n_type * n_type * n_coeff; p += blockDim.x )
        coeffs[p] = coefficients[p];
    __syncthreads();

    for ( int iter = id_in_partition; ; iter += part_size)
    {
        int i = (p_beg & WARPALIGN) + iter;
        if (i >= p_end) break;
        if (i >= p_beg)
        {
            f3u coord1 = tex1Dfetch<float4>( tex_coord, i);
            f3u veloc1 = tex1Dfetch<float4>( tex_veloc, i);
            int n_pair = pair_count[i];
            int *p_pair = pair_table + ( i - __laneid() + part_id ) * pair_padding + __laneid();
            r64 fx = 0., fy = 0., fz = 0.;
            r64 vrxx = 0., vryy = 0., vrzz = 0.;
            r64 vrxy = 0., vrxz = 0., vryz = 0.;
            r64 energy = 0.;

            for ( int p = part_id; p < n_pair; p += n_part)
            {
                int j = __lds( p_pair );
                p_pair += pair_padding * n_part;
                if ( (p & 31) + n_part >= WARPSZ ) p_pair -= WARPSZ * pair_padding - WARPSZ;

                f3u coord2 = tex1Dfetch<float4>( tex_coord, j);
                r64 dx = coord1.x - coord2.x;
                r64 dy = coord1.y - coord2.y;
                r64 dz = coord1.z - coord2.z;
                r64 rsq = dx * dx + dy * dy + dz * dz;
                r64 *coeff_ij = coeffs + ( coord1.i * n_type + coord2.i ) * n_coeff;

                if ( rsq < coeff_ij[p_cutsq] && rsq >= EPSILON_SQ)
                {
                    f3u veloc2 = tex1Dfetch<float4>( tex_veloc, j);
                    r64 r2inv = 1.0/rsq;
                    r64 r6inv = r2inv * r2inv * r2inv;
                    r64 forcelj = r6inv * (coeff_ij[p_lj1] * r6inv - coeff_ij[p_lj2]);

                    r64 fpair = forcelj * r2inv;
                    fx += fpair * dx;
                    fy += fpair * dy;
                    fz += fpair * dz;

                    if ( evflag )
                    {
                        vrxx += dx * dx * fpair;
                        vryy += dy * dy * fpair;
                        vrzz += dz * dz * fpair;
                        vrxy += dx * dy * fpair;
                        vrxz += dx * dz * fpair;
                        vryz += dy * dz * fpair;
                        energy = r6inv * (coeff_ij[p_lj3] * r6inv - coeff_ij[p_lj4]) - coeff_ij[p_offset];
                    }
                }
            }

            if ( n_part == 1)
            {
                force_x[i] += fx;
                force_y[i] += fy;
                force_z[i] += fz;
                
                if (evflag)
                {
                    virial_xx[i] += vrxx * 0.5;
                    virial_yy[i] += vryy * 0.5;
                    virial_zz[i] += vrzz * 0.5;
                    virial_xy[i] += vrxy * 0.5;
                    virial_xz[i] += vrxz * 0.5;
                    virial_yz[i] += vryz * 0.5;
                    e_pair[i] = energy * 0.5;
                }
            }
            else
            {
                atomic_add( force_x + i, fx);
                atomic_add( force_y + i, fy);
                atomic_add( force_z + i, fz);
                
                if (evflag)
                {
                    atomic_add( virial_xx + i, vrxx * 0.5 );
                    atomic_add( virial_yy + i, vryy * 0.5 );
                    atomic_add( virial_zz + i, vrzz * 0.5 );
                    atomic_add( virial_xy + i, vrxy * 0.5 );
                    atomic_add( virial_xz + i, vrxz * 0.5 );
                    atomic_add( virial_yz + i, vryz * 0.5 );
                    atomic_add( e_pair + i, energy * 0.5);
                }
            }
        }
    }
}

void MesoPairMD::compute_kernel( int eflag, int vflag, int p_beg, int p_end )
{
    if ( !coeff_ready ) prepare_coeff();
    MesoNeighList *dlist = meso_neighbor->lists_device[ list-> index ];

    int shared_mem_size = atom->ntypes * atom->ntypes * n_coeff * sizeof( r64 );

    if ( eflag || vflag )
    {
        // evaluate force, energy and virial
        static GridConfig grid_cfg = meso_device->configure_kernel( gpu_pair_md<1>, shared_mem_size );
        gpu_pair_md<1> <<< grid_cfg.x, grid_cfg.y, shared_mem_size, meso_device->stream() >>>(
            meso_atom->tex_coord_merged, meso_atom->tex_veloc_merged,
            meso_atom->dev_force(0), meso_atom->dev_force(1), meso_atom->dev_force(2),
            meso_atom->dev_virial(0), meso_atom->dev_virial(1), meso_atom->dev_virial(2),
            meso_atom->dev_virial(3), meso_atom->dev_virial(4), meso_atom->dev_virial(5),
            dlist->dev_pair_count_core, dlist->dev_pair_table, meso_atom->dev_e_pair,
            dev_coefficients, dlist->n_col, atom->ntypes,
            p_beg, p_end, grid_cfg.partition( p_end - p_beg, WARPSZ)
        );
    }
    else
    {
        // evaluate force only
        static GridConfig grid_cfg = meso_device->configure_kernel( gpu_pair_md<0>, shared_mem_size );
        gpu_pair_md<0> <<< grid_cfg.x, grid_cfg.y, shared_mem_size, meso_device->stream() >>>(
            meso_atom->tex_coord_merged, meso_atom->tex_veloc_merged,
            meso_atom->dev_force(0), meso_atom->dev_force(1), meso_atom->dev_force(2),
            meso_atom->dev_virial(0), meso_atom->dev_virial(1), meso_atom->dev_virial(2),
            meso_atom->dev_virial(3), meso_atom->dev_virial(4), meso_atom->dev_virial(5),
            dlist->dev_pair_count_core, dlist->dev_pair_table, meso_atom->dev_e_pair,
            dev_coefficients, dlist->n_col, atom->ntypes,
            p_beg, p_end, grid_cfg.partition( p_end - p_beg, WARPSZ)
        );
    }
}

void MesoPairMD::compute_bulk( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::BULK, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::LOCAL, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}

void MesoPairMD::compute_border( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::BORDER, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::GHOST, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}

void MesoPairMD::compute( int eflag, int vflag )
{
    int p_beg, p_end, c_beg, c_end;
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::LOCAL, p_beg, p_end );
    meso_atom->meso_avec->resolve_work_range( AtomAttribute::ALL, c_beg, c_end );
    meso_atom->meso_avec->dp2sp_merged( seed_now(), c_beg, c_end, true ); // convert coordinates data to r32
    compute_kernel( eflag, vflag, p_beg, p_end );
}


uint MesoPairMD::seed_now()
{
    return premix_TEA<64>( seed, update->ntimestep );
}

void MesoPairMD::settings( int narg, char **arg )
{
    if ( narg < 1) error->all( FLERR, "Illegal pair_style command" );

    cut_global = atof( arg[0] );
    if ( narg > 1) seed = atoi( arg[1] );

    // reset cutoffs that have been explicitly set
    if ( allocated )
    {
        for ( int i = 1; i <= atom->ntypes; i++ )
            for ( int j = 1; j <= atom->ntypes; j++ )
                if ( setflag[i][j] )
                    cut[i][j] = cut_global;
    }
}

void MesoPairMD::coeff( int narg, char **arg )
{
    if ( narg <4 || narg > 5)
        error->all( FLERR, "Incorrect args for pair coefficients" );
    if ( !allocated ) allocate();

    int ilo, ihi, jlo, jhi;
    force->bounds( arg[0], atom->ntypes, ilo, ihi );
    force->bounds( arg[1], atom->ntypes, jlo, jhi );

    double epsilon_one = atof( arg[2] );
    double sigma_one = atof( arg[3] );
    double cut_one = cut_global;
    if ( narg == 5 ) cut_one = atof( arg[4] );

    int count = 0;
    for ( int i = ilo; i <= ihi; i++)
    {
        for ( int j = MAX( jlo, i ); j <= jhi; j++)
        {
            epsilon[i][j] = epsilon_one;
            sigma[i][j] = sigma_one;
            cut[i][j] = cut_one;
            cutsq[i][j] = cut_one * cut_one;
            setflag[i][j] = 1;
            count++;
        }
    }

    if ( count == 0 ) error->all( FLERR, "Incorrect args for pair coefficients" );
}

/* ----------------------------------------------------------------------
 init specific to MD pair style
 ------------------------------------------------------------------------- */

void MesoPairMD::init_style()
{
    int i = neighbor->request( this );
    neighbor->requests[i]->cudable = 1;
    neighbor->requests[i]->newton = 2;
}

/* ----------------------------------------------------------------------
 init for one type pair i,j and corresponding j,i
 ------------------------------------------------------------------------- */

double MesoPairMD::init_one( int i, int j )
{
    if ( setflag[i][j] == 0 )
    {
        epsilon[i][j] = mix_energy( epsilon[i][i],epsilon[j][j],epsilon[i][i],sigma[j][j] );
        sigma[i][j] = mix_distance( sigma[i][i], sigma[j][j] );
        cut[i][j] = mix_distance( cut[i][i], cut[j][j] );
    }

    lj1[i][j] = 48.0 * epsilon[i][j] * pow( sigma[i][j], 12.0 );
    lj2[i][j] = 24.0 * epsilon[i][j] * pow( sigma[i][j], 6.0 );
    lj3[i][j] = 4.0 * epsilon[i][j] * pow( sigma[i][j], 12.0 );
    lj4[i][j] = 4.0 * epsilon[i][j] * pow( sigma[i][j], 6.0 );
    
    if ( offset_flag ){
        double ratio = sigma[i][j] / cut[i][j];
        offset[i][j] = 4.0 * epsilon[i][j] * ( pow(ratio, 12.0) - pow(ratio, 6.0));
    } else offset[i][j] = 0.0;

    lj1[j][i] = lj1[i][j];
    lj2[j][i] = lj2[i][j];
    lj3[j][i] = lj3[i][j];
    lj4[j][i] = lj4[i][j];
    offset[j][i] = offset[i][j];

    if (tail_flag) {
        int *type = atom->type;
        int nlocal = atom->nlocal;

        double count[2],all[2];
        count[0] = count[1] = 0.0;

        for (int k = 0; k < nlocal; k++) {
            if (type[k] == i) count[0] += 1.0;
            if (type[k] == j) count[1] += 1.0;
        }
        
        MPI_Allreduce(count,all,2,MPI_DOUBLE,MPI_SUM,world);

        double sig2 = sigma[i][j]*sigma[i][j];
        double sig6 = sig2*sig2*sig2;
        double rc3 = cut[i][j]*cut[i][j]*cut[i][j];
        double rc6 = rc3*rc3;
        double rc9 = rc3*rc6;
        etail_ij = 8.0*MY_PI*all[0]*all[1]*epsilon[i][j] * sig6 * (sig6 - 3.0*rc6) / (9.0*rc9);
        ptail_ij = 16.0*MY_PI*all[0]*all[1]*epsilon[i][j] * sig6 * (2.0*sig6 - 3.0*rc6) / (9.0*rc9);
    }

    return cut[i][j];
}

/* ----------------------------------------------------------------------
   proc 0 writes to restart file
------------------------------------------------------------------------- */

void MesoPairMD::write_restart(FILE *fp)
{
    write_restart_settings(fp);

    for ( int i = 1; i <= atom->ntypes; i++)
    {
        for ( int j = i; j <= atom->ntypes; j++) 
        {
            fwrite( &setflag[i][j], sizeof(int), 1, fp );
            if (setflag[i][j]) 
            {
                fwrite( &epsilon[i][j], sizeof(double), 1, fp);
                fwrite( &sigma[i][j], sizeof(double), 1, fp);
                fwrite( &cut[i][j], sizeof(double), 1, fp);
            }
        }
    }
}

/* ----------------------------------------------------------------------
   proc 0 reads from restart file, bcasts
------------------------------------------------------------------------- */

void MesoPairMD::read_restart(FILE *fp)
{
    read_restart_settings( fp );
    allocate();

    int me = comm->me;
    for ( int i = 1; i <= atom->ntypes; i++ )
    {
        for ( int j = i; j <= atom->ntypes; j++ )
        {
            if (me == 0) fread(&setflag[i][j],sizeof(int),1,fp);
            MPI_Bcast(&setflag[i][j],1,MPI_INT,0,world);

            if (setflag[i][j]) 
            {
                if (me == 0) 
                {
                    fread( &epsilon[i][j], sizeof(double), 1, fp);
                    fread( &sigma[i][j], sizeof(double), 1, fp);
                    fread( &cut[i][j], sizeof(double), 1, fp);
                }
                MPI_Bcast( &epsilon[i][j], 1, MPI_DOUBLE, 0, world);
                MPI_Bcast( &sigma[i][j], 1, MPI_DOUBLE, 0, world);
                MPI_Bcast( &cut[i][j], 1, MPI_DOUBLE, 0, world);
            }
        }
    }
}

/* ----------------------------------------------------------------------
 proc 0 writes to restart file
 ------------------------------------------------------------------------- */

void MesoPairMD::write_restart_settings( FILE *fp )
{
    fwrite( &cut_global, sizeof( double ), 1, fp );
    fwrite( &offset_flag, sizeof( int ), 1, fp);
    fwrite( &seed, sizeof( int ), 1, fp );
    fwrite( &mix_flag, sizeof( int ), 1, fp );
}

/* ----------------------------------------------------------------------
 proc 0 reads from restart file, bcasts
 ------------------------------------------------------------------------- */

void MesoPairMD::read_restart_settings( FILE *fp )
{
    if( comm->me == 0 ) {
        fread( &cut_global, sizeof( double ), 1, fp );
        fread( &offset_flag, sizeof( int ), 1, fp);
        fread( &seed, sizeof( int ), 1, fp );
        fread( &mix_flag, sizeof( int ), 1, fp );
    }
    MPI_Bcast( &cut_global, 1, MPI_DOUBLE, 0, world );
    MPI_Bcast( &offset_flag, 1, MPI_INT, 0, world );
    MPI_Bcast( &seed, 1, MPI_INT, 0, world );
    MPI_Bcast( &mix_flag, 1, MPI_INT, 0, world );
}

double MesoPairMD::single( int i, int j, int itype, int jtype, double rsq,
                           double factor_coul, double factor_lj, double &fforce)
{
    double r2inv, r6inv, forcelj, philj;

    r2inv = 1.0 / rsq;
    r6inv = r2inv * r2inv * r2inv;
    forcelj = r6inv * (lj1[itype][jtype] * r6inv - lj2[itype][jtype]);
    fforce = factor_lj * forcelj * r2inv; 

    philj = r6inv * (lj3[itype][jtype]*r6inv-lj4[itype][jtype]) - offset[itype][jtype];
    return factor_lj * philj;
}
